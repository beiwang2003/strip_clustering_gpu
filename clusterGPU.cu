#include "hip/hip_runtime.h"
#include "clusterGPU.cuh"
#include <hipcub/hipcub.hpp>
#include <stdio.h>

#ifdef USE_TEXTURE
texture<float, 1, hipReadModeElementType> noiseTexRef;
texture<float, 1, hipReadModeElementType> gainTexRef;
texture<uint16_t, 1, hipReadModeElementType> stripIdTexRef;
texture<uint16_t, 1, hipReadModeElementType> adcTexRef;

static __inline__ __device__ float fetch_noise(int i)
{
  return tex1Dfetch(noiseTexRef, i);
}
static __inline__ __device__ float fetch_gain(int i)
{
  return tex1Dfetch(gainTexRef, i);
}
static __inline__ __device__ uint16_t fetch_stripId(int i)
{
  return tex1Dfetch(stripIdTexRef, i);
}

static __inline__ __device__ uint16_t fetch_adc(int i)
{
  return tex1Dfetch(adcTexRef, i);
}
#define NOISE(i) (fetch_noise(i))
#define GAIN(i) (fetch_gain(i))
#define STRIPID(i) (fetch_stripId(i))
#define ADC(i) (fetch_adc(i))
#else
#define NOISE(i) (noise_d[i])
#define GAIN(i) (gain_d[i])
#define STRIPID(i) (stripId_d[i])
#define ADC(i) (adc_d[i])
#endif

static void gpu_timer_start(gpu_timing_t *gpu_timing)
{
  hipEventCreate(&gpu_timing->start);
  hipEventCreate(&gpu_timing->stop);
  hipEventRecord(gpu_timing->start,0);
}

static float gpu_timer_measure(gpu_timing_t *gpu_timing)
{
  float elapsedTime;
  hipEventRecord(gpu_timing->stop,0);
  hipEventSynchronize(gpu_timing->stop);
  hipEventElapsedTime(&elapsedTime, gpu_timing->start, gpu_timing->stop);
  hipEventRecord(gpu_timing->start,0);

  return elapsedTime/1000;
}

static float gpu_timer_measure_end(gpu_timing_t *gpu_timing)
{
  float elapsedTime;
  hipEventRecord(gpu_timing->stop,0);
  hipEventSynchronize(gpu_timing->stop);
  hipEventElapsedTime(&elapsedTime, gpu_timing->start,gpu_timing->stop);

  hipEventDestroy(gpu_timing->start);
  hipEventDestroy(gpu_timing->stop);
  return elapsedTime/1000;
}


__global__
static void setSeedStripsGPU(int nStrips, float *noise_d, uint16_t *adc_d, int *seedStripMask_d, int *seedStripNCMask_d)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;
  float SeedThreshold = 3.0;

  int i = nthreads * bid + tid;

  if (i<nStrips) {
    seedStripMask_d[i] = 0;
    seedStripNCMask_d[i] = 0;
    float noise_i = NOISE(i);
    uint8_t adc_i = static_cast<uint8_t>(ADC(i));
    seedStripMask_d[i] = (adc_i >= static_cast<uint8_t>( noise_i * SeedThreshold)) ? 1:0;
    seedStripNCMask_d[i] = seedStripMask_d[i];
  }
}

__global__
static void setNCSeedStripsGPU(int nStrips, uint16_t *stripId_d, int *seedStripMask_d, int *seedStripNCMask_d)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;

  int i = nthreads * bid + tid;

  if (i>0&&i<nStrips) {
    if (seedStripMask_d[i]&&seedStripMask_d[i-1]&&(STRIPID(i)-STRIPID(i-1))==1) seedStripNCMask_d[i] = 0;
  }
}

__global__
static void setStripIndexGPU(int nStrips, int *seedStripNCMask_d, int *prefixSeedStripNCMask_d, int *seedStripsNCIndex_d)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;

  int i = nthreads * bid + tid;

  if (i<nStrips) {
    if (seedStripNCMask_d[i] == 1) {
      int index = prefixSeedStripNCMask_d[i];
      seedStripsNCIndex_d[index] = i;
    }
  }

}

__global__
static void findLeftRightBoundaryGPU(int nSeedStripsNC, int nStrips, float* clusterNoiseSquared_d, int *clusterLastIndexLeft_d, int *clusterLastIndexRight_d, int *seedStripsNCIndex_d, uint16_t *stripId_d, uint16_t *adc_d, float *noise_d)
{
   const uint8_t MaxSequentialHoles = 0;
   const float  ChannelThreshold = 2.0;
   const int tid = threadIdx.x;
   const int bid = blockIdx.x;
   const int nthreads = blockDim.x;

   int index, testIndexLeft, testIndexRight, indexLeft, indexRight, rangeLeft, rangeRight;
   uint8_t testADC;
   float noise_i, testNoise;

   int i = nthreads * bid + tid;

   if (i<nSeedStripsNC) {

     clusterNoiseSquared_d[i] = 0.0;
     index=seedStripsNCIndex_d[i];
     indexLeft = index;
     indexRight = index;
     noise_i = NOISE(index);
     //noise_i = noise_d[index];
     clusterNoiseSquared_d[i] += noise_i*noise_i;

     // find left boundary
     testIndexLeft=index-1;
     rangeLeft = STRIPID(indexLeft)-STRIPID(testIndexLeft)-1;
     //rangeLeft = stripId_d[indexLeft]-stripId_d[testIndexLeft]-1;
     while(testIndexLeft>=0&&rangeLeft>=0&&rangeLeft<=MaxSequentialHoles) {
       //testNoise = NOISE(testIndexLeft);
       //testADC = static_cast<uint8_t>(ADC(testIndexLeft));
       testNoise = noise_d[testIndexLeft];
       testADC = static_cast<uint8_t>(adc_d[testIndexLeft]);
       if (testADC > static_cast<uint8_t>(testNoise * ChannelThreshold)) {
	 --indexLeft;
	 clusterNoiseSquared_d[i] += testNoise*testNoise;
       }
       --testIndexLeft;
       rangeLeft = STRIPID(indexLeft)-STRIPID(testIndexLeft)-1;
       //rangeLeft = stripId_d[indexLeft]-stripId_d[testIndexLeft]-1;
     }

     // find right boundary
     testIndexRight=index+1;
     rangeRight = STRIPID(testIndexRight)-STRIPID(indexRight)-1;
     //rangeRight = stripId_d[testIndexRight]-stripId_d[indexRight]-1;
     while(testIndexRight<nStrips&&rangeRight>=0&&rangeRight<=MaxSequentialHoles) {
       //testNoise = NOISE(testIndexRight);
       //testADC = static_cast<uint8_t>(ADC(testIndexRight));
       testNoise = noise_d[testIndexRight];
       testADC = static_cast<uint8_t>(adc_d[testIndexRight]);
       if (testADC > static_cast<uint8_t>(testNoise * ChannelThreshold)) {
	 ++indexRight;
	 clusterNoiseSquared_d[i] += testNoise*testNoise;
       }
       ++testIndexRight;
       rangeRight = STRIPID(testIndexRight)-STRIPID(indexRight)-1;
       //rangeRight= stripId_d[testIndexRight]-stripId_d[indexRight]-1;
     }

     clusterLastIndexLeft_d[i] = indexLeft;
     clusterLastIndexRight_d[i] = indexRight;
   }
}

__global__
static void checkClusterConditionGPU(int nSeedStripsNC, float* clusterNoiseSquared_d, int *clusterLastIndexLeft_d, int *clusterLastIndexRight_d, uint16_t *adc_d, float * gain_d, bool *trueCluster_d, uint8_t *clusterADCs_d)
{
   const float minGoodCharge = 1620.0;
   const float ClusterThresholdSquared = 25.0;

   const int tid = threadIdx.x;
   const int bid = blockIdx.x;
   const int nthreads = blockDim.x;

   int i = nthreads * bid + tid;

   int left, right, size, j;
   int adcsum = 0, charge;
   bool noiseSquaredPass, chargePerCMPass;
   uint8_t adc_j;
   float gain_j;

   if (i<nSeedStripsNC) {

    trueCluster_d[i] = false;
    left=clusterLastIndexLeft_d[i];
    right=clusterLastIndexRight_d[i];
    size=right-left+1;
    for (j=0; j<size; j++) {
      adcsum += (int)ADC(left+j);
    }
    noiseSquaredPass = clusterNoiseSquared_d[i]*ClusterThresholdSquared <= ((float)(adcsum)*float(adcsum));
    chargePerCMPass = (float)(adcsum)/0.047f > minGoodCharge;
    if (noiseSquaredPass&&chargePerCMPass) {
      for (j=0; j<size; j++){
        adc_j = ADC(left+j);
        gain_j = GAIN(left+j);
        charge = int( float(adc_j)/gain_j + 0.5f );
        if (adc_j < 254) adc_j = ( charge > 1022 ? 255 : (charge > 253 ? 254 : charge));
        clusterADCs_d[j*nSeedStripsNC+i] = adc_j;
      }
      trueCluster_d[i] = true;
    }
  }
}

extern "C"
void allocateMemAllStripsGPU(int max_strips, uint16_t **stripId_d_pt, uint16_t **adc_d_pt, float **noise_d_pt, float **gain_d_pt, int **seedStripsNCIndex_d_pt)
{
  hipMalloc(stripId_d_pt, max_strips*sizeof(uint16_t));
  hipMalloc(adc_d_pt, max_strips*sizeof(uint16_t));
  hipMalloc(noise_d_pt, max_strips*sizeof(float));
  hipMalloc(gain_d_pt, max_strips*sizeof(float));
  hipMalloc(seedStripsNCIndex_d_pt, max_strips*sizeof(int));
}

extern "C"
void allocateMemNCSeedStripsGPU(int nSeedStripsNC, int **clusterLastIndexLeft_d_pt, int **clusterLastIndexRight_d_pt, float **clusterNoiseSquared_d_pt, uint8_t **clusterADCs_d_pt, bool **trueCluster_d_pt)
{
  hipMalloc(clusterLastIndexLeft_d_pt, 2*nSeedStripsNC*sizeof(int));
  *clusterLastIndexRight_d_pt = *clusterLastIndexLeft_d_pt + nSeedStripsNC;
  hipMalloc(clusterNoiseSquared_d_pt, nSeedStripsNC*sizeof(float));
  hipMalloc(clusterADCs_d_pt, nSeedStripsNC*256*sizeof(uint8_t));
  hipMalloc(trueCluster_d_pt, nSeedStripsNC*sizeof(bool));
}

extern "C"
void freeGPUMem(uint16_t *stripId_d, uint16_t *adc_d, float *noise_d, float *gain_d, int *seedStripNCIndex_d, int *clusterLastIndexLeft_d, float *clusterNoiseSquared_d, uint8_t *clusterADCs_d, bool *trueCluster_d)
{
   hipFree(stripId_d);
   hipFree(adc_d);
   hipFree(noise_d);
   hipFree(gain_d);
#ifdef USE_TEXTURE
   hipUnbindTexture(stripIdTexRef);
   hipUnbindTexture(adcTexRef);
   hipUnbindTexture(noiseTexRef);
   hipUnbindTexture(gainTexRef);
#endif
   hipFree(seedStripNCIndex_d);
   hipFree(clusterLastIndexLeft_d);
   hipFree(clusterNoiseSquared_d);
   hipFree(clusterADCs_d);
   hipFree(trueCluster_d);
}

extern "C"
void  findClusterGPU(int nSeedStripsNC, int nStrips, float *clusterNoiseSquared_d, int *clusterLastIndexLeft_d,  int *clusterLastIndexRight_d, int *seedStripsNCIndex_d, uint16_t *stripId_d, uint16_t *adc_d, float *noise_d, float *gain_d, bool *trueCluster_d, uint8_t *clusterADCs_d, gpu_timing_t *gpu_timing)
{
  gpu_timer_start(gpu_timing);
  int nthreads = 256;
  int nblocks = (nSeedStripsNC+nthreads-1)/nthreads;

#ifdef GPU_DEBUG
  int *cpu_index = (int *)malloc(nSeedStripsNC*sizeof(int));
  uint16_t *cpu_strip = (uint16_t *)malloc(nStrips*sizeof(uint16_t));
  uint16_t *cpu_adc = (uint16_t *)malloc(nStrips*sizeof(uint16_t));
  float *cpu_noise = (float *)malloc(nStrips*sizeof(float));

  hipMemcpy((void *)cpu_strip, stripId_d, nStrips*sizeof(uint16_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_adc, adc_d, nStrips*sizeof(uint16_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_noise, noise_d, nStrips*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_index, seedStripsNCIndex_d, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);

  for (int i=0; i<nStrips; i++) {
    std::cout<<" cpu_strip "<<cpu_strip[i]<<" cpu_adc "<<cpu_adc[i]<<" cpu_noise "<<cpu_noise[i]<<" cpu index "<<cpu_index[i]<<std::endl;
  }

  int *clusterL_d, *clusterR_d;
  float *clusterNoiseSq_d;
  bool *trueC_d;
  uint8_t *cADCs_d;
  hipMalloc(&clusterL_d, nSeedStripsNC*sizeof(int));
  hipMalloc(&clusterR_d, nSeedStripsNC*sizeof(int));
  hipMalloc(&clusterNoiseSq_d, nSeedStripsNC*sizeof(float));
  hipMalloc(&trueC_d, nSeedStripsNC*sizeof(bool));
  hipMalloc(&cADCs_d, nSeedStripsNC*256*sizeof(uint8_t));
#endif

  findLeftRightBoundaryGPU<<<nblocks, nthreads>>>(nSeedStripsNC, nStrips, clusterNoiseSquared_d, clusterLastIndexLeft_d, clusterLastIndexRight_d, seedStripsNCIndex_d, stripId_d, adc_d, noise_d);
  //findLeftRightBoundaryGPU<<<nblocks, nthreads>>>(nSeedStripsNC, nStrips, clusterNoiseSq_d, clusterL_d, clusterR_d, seedStripsNCIndex_d, stripId_d, adc_d, noise_d);

  gpu_timing->findBoundaryTime = gpu_timer_measure(gpu_timing);

  //hipDeviceSynchronize();

  checkClusterConditionGPU<<<nblocks, nthreads>>>(nSeedStripsNC, clusterNoiseSquared_d, clusterLastIndexLeft_d, clusterLastIndexRight_d, adc_d, gain_d, trueCluster_d, clusterADCs_d);
  //checkClusterConditionGPU<<<nblocks, nthreads>>>(nSeedStripsNC, clusterNoiseSq_d, clusterL_d, clusterR_d, adc_d, gain_d, trueC_d, cADCs_d);

  //hipDeviceSynchronize();
  gpu_timing->checkClusterTime = gpu_timer_measure_end(gpu_timing);

#ifdef GPU_DEBUG
  int *clusterLastIndexLeft = (int *)malloc(nSeedStripsNC*sizeof(int));
  int *clusterLastIndexRight = (int *)malloc(nSeedStripsNC*sizeof(int));
  bool *trueCluster = (bool *)malloc(nSeedStripsNC*sizeof(bool));
  uint8_t *ADCs = (uint8_t*)malloc(nSeedStripsNC*256*sizeof(uint8_t));


  hipMemcpy((void *)clusterLastIndexLeft, clusterL_d, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)clusterLastIndexRight, clusterR_d, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)trueCluster, trueC_d, nSeedStripsNC*sizeof(bool), hipMemcpyDeviceToHost);
  hipMemcpy((void *)ADCs, cADCs_d, nSeedStripsNC*256*sizeof(uint8_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)clusterLastIndexLeft, clusterLastIndexLeft_d, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)clusterLastIndexRight, clusterLastIndexRight_d, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)trueCluster, trueCluster_d, nSeedStripsNC*sizeof(bool), hipMemcpyDeviceToHost);
  hipMemcpy((void *)ADCs, clusterADCs_d, nSeedStripsNC*256*sizeof(uint8_t), hipMemcpyDeviceToHost);

  /*
  for (int i=0; i<nSeedStripsNC; i++) {
    if (trueCluster[i]){
      int left=clusterLastIndexLeft[i];
      int right=clusterLastIndexRight[i];
      std::cout<<" left "<<left<<" right "<<right<<" : ";
      int size=right-left+1;
      for (int j=0; j<size; j++){
	std::cout<<(int)ADCs[j*nSeedStripsNC+i]<<" ";
      }
      std::cout<<std::endl;
    }
  }
  */

  free(cpu_index);
  free(cpu_strip);
  free(cpu_adc);
  free(cpu_noise);

  free(clusterLastIndexLeft);
  free(clusterLastIndexRight);
  free(trueCluster);
  free(ADCs);

  hipFree(clusterL_d);
  hipFree(clusterR_d);
  hipFree(clusterNoiseSq_d);
  hipFree(trueC_d);
  hipFree(cADCs_d);
#endif

}

extern "C"
int setSeedStripsNCIndexGPU(int nStrips, uint16_t *stripId_d, uint16_t *adc_d, float *noise_d, int *seedStripsNCIndex_d, gpu_timing_t *gpu_timing){
  int nSeedStripsNC;

  gpu_timer_start(gpu_timing);

  int *seedStripsMask_d, *seedStripsNCMask_d, *prefixSeedStripsNCMask_d;
  hipMalloc((void **)&seedStripsMask_d, 3*nStrips*sizeof(int));
  seedStripsNCMask_d = seedStripsMask_d + nStrips;
  prefixSeedStripsNCMask_d = seedStripsMask_d + 2*nStrips;

  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;

#ifdef GPU_DEBUG
  int *cpu_mask = (int *)malloc(nStrips*sizeof(int));
  int *cpu_prefix= (int *)malloc(nStrips*sizeof(int));
  int *cpu_index = (int *)malloc(nStrips*sizeof(int));
#endif

  int nthreads = 256;
  int nblocks = (nStrips+nthreads-1)/nthreads;

  // mark seed strips
  setSeedStripsGPU<<<nblocks, nthreads>>>(nStrips, noise_d, adc_d, seedStripsMask_d, seedStripsNCMask_d);
  gpu_timing->setSeedStripsTime = gpu_timer_measure(gpu_timing);


  // mark only non-consecutive seed strips (mask out consecutive seed strips)
  setNCSeedStripsGPU<<<nblocks, nthreads>>>(nStrips, stripId_d, seedStripsMask_d, seedStripsNCMask_d);
  gpu_timing->setNCSeedStripsTime = gpu_timer_measure(gpu_timing);

  // set index for non-consecutive seed strips
  hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, seedStripsNCMask_d, prefixSeedStripsNCMask_d, nStrips);

  hipMalloc(&d_temp_storage, temp_storage_bytes);

  hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, seedStripsNCMask_d, prefixSeedStripsNCMask_d, nStrips);

  hipMemcpy((void *)&nSeedStripsNC, prefixSeedStripsNCMask_d+nStrips-1, sizeof(int), hipMemcpyDeviceToHost);

  setStripIndexGPU<<<nblocks, nthreads>>>(nStrips, seedStripsNCMask_d, prefixSeedStripsNCMask_d, seedStripsNCIndex_d);

  hipFree(d_temp_storage);

#ifdef GPU_DEBUG
  hipMemcpy((void *)cpu_mask, seedStripsNCMask_d, nStrips*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_prefix, prefixSeedStripsNCMask_d, nStrips*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_index, seedStripsNCIndex_d, nStrips*sizeof(int), hipMemcpyDeviceToHost);

  for (int i=0; i<nStrips; i++) {
    std::cout<<" i "<<i<<" mask "<<cpu_mask[i]<<" prefix "<<cpu_prefix[i]<<" index "<<cpu_index[i]<<std::endl;
  }

  free(cpu_mask);
  free(cpu_prefix);
  free(cpu_index);

  std::cout<<"nStrips="<<nStrips<<"nSeedStripsNC="<<nSeedStripsNC<<"temp_storage_bytes="<<temp_storage_bytes<<std::endl;
#endif

  hipFree(seedStripsMask_d);

  gpu_timing->setStripIndexTime = gpu_timer_measure_end(gpu_timing);

  return nSeedStripsNC;
}

extern "C"
void cpyGPUToCPU(int nSeedStripsNC, int *clusterLastIndexLeft_d, int *clusterLastIndexLeft, int *clusterLastIndexRight_d, int *clusterLastIndexRight, uint8_t *clusterADCs_d, uint8_t *clusterADCs, bool *trueCluster_d, bool *trueCluster) {
  hipMemcpy((void *)clusterLastIndexLeft, clusterLastIndexLeft_d, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)clusterLastIndexRight, clusterLastIndexRight_d, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)clusterADCs, clusterADCs_d, nSeedStripsNC*256*sizeof(uint8_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)trueCluster, trueCluster_d, nSeedStripsNC*sizeof(bool), hipMemcpyDeviceToHost);
}

extern "C"
void cpyCPUToGPU(int nStrips, uint16_t *stripId_d, uint16_t *stripId, uint16_t *adc_d,  uint16_t *adc, float *noise_d, float *noise, float *gain_d, float *gain, gpu_timing_t *gpu_timing) {
  gpu_timer_start(gpu_timing);
  hipMemcpy((void *)stripId_d, stripId, nStrips*sizeof(uint16_t), hipMemcpyHostToDevice);
  hipMemcpy((void *)adc_d, adc, nStrips*sizeof(uint16_t), hipMemcpyHostToDevice);
  hipMemcpy((void *)noise_d, noise, nStrips*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy((void *)gain_d, gain, nStrips*sizeof(float), hipMemcpyHostToDevice);
#ifdef USE_TEXTURE
  hipBindTexture(0, stripIdTexRef, (void *)stripId_d, nStrips*sizeof(uint16_t));
  hipBindTexture(0, adcTexRef, (void *)adc_d, nStrips*sizeof(uint16_t));
  hipBindTexture(0, noiseTexRef, (void *)noise_d, nStrips*sizeof(float));
  hipBindTexture(0, gainTexRef, (void *)gain_d, nStrips*sizeof(float));
#endif
  gpu_timing->memTransferTime = gpu_timer_measure_end(gpu_timing);
}
