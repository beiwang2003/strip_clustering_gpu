#include "hip/hip_runtime.h"
#include "clusterGPU.cuh"
#include <stdio.h>
#include <hipcub/hipcub.hpp>
#include <cub/util_allocator.cuh>
#ifdef CACHE_ALLOC
#include "allocate_device.h"
#endif
#include "copyAsync.h"

constexpr auto kStripsPerChannel = SiStripConditionsBase::kStripsPerChannel;

#if USE_TEXTURE
texture<float, 1, hipReadModeElementType> noiseTexRef;
texture<float, 1, hipReadModeElementType> gainTexRef;
texture<uint16_t, 1, hipReadModeElementType> stripIdTexRef;
texture<uint8_t, 1, hipReadModeElementType> adcTexRef;

static __inline__ __device__ float fetch_noise(int i)
{
  return tex1Dfetch(noiseTexRef, i);
}
static __inline__ __device__ float fetch_gain(int i)
{
  return tex1Dfetch(gainTexRef, i);
}
static __inline__ __device__ uint16_t fetch_stripId(int i)
{
  return tex1Dfetch(stripIdTexRef, i);
}

static __inline__ __device__ uint8_t fetch_adc(int i)
{
  return tex1Dfetch(adcTexRef, i);
}
#define NOISE(i) (fetch_noise(i))
#define GAIN(i) (fetch_gain(i))
#define STRIPID(i) (fetch_stripId(i))
#define ADC(i) (fetch_adc(i))
#else
#define NOISE(i) (noise[i])
#define GAIN(i) (gain[i])
#define STRIPID(i) (stripId[i])
#define ADC(i) (adc[i])
#endif

static void gpu_timer_start(gpu_timing_t *gpu_timing, hipStream_t stream) {
  CUDA_RT_CALL(hipEventCreate(&gpu_timing->start));
  CUDA_RT_CALL(hipEventCreate(&gpu_timing->stop));
  CUDA_RT_CALL(hipEventRecord(gpu_timing->start, stream));
}

static float gpu_timer_measure(gpu_timing_t *gpu_timing, hipStream_t stream) {
  float elapsedTime;
  CUDA_RT_CALL(hipEventRecord(gpu_timing->stop, stream));
  CUDA_RT_CALL(hipEventSynchronize(gpu_timing->stop));
  CUDA_RT_CALL(hipEventElapsedTime(&elapsedTime, gpu_timing->start, gpu_timing->stop));
  CUDA_RT_CALL(hipEventRecord(gpu_timing->start, stream));

  return elapsedTime/1000;
}

static float gpu_timer_measure_end(gpu_timing_t *gpu_timing, hipStream_t stream) {
  float elapsedTime;
  CUDA_RT_CALL(hipEventRecord(gpu_timing->stop,stream));
  CUDA_RT_CALL(hipEventSynchronize(gpu_timing->stop));
  CUDA_RT_CALL(hipEventElapsedTime(&elapsedTime, gpu_timing->start,gpu_timing->stop));

  CUDA_RT_CALL(hipEventDestroy(gpu_timing->start));
  CUDA_RT_CALL(hipEventDestroy(gpu_timing->stop));
  return elapsedTime/1000;
}

__global__
static void unpackChannelsSST(const ChanLocStruct* chanlocs, const SiStripConditionsGPU* conditions, sst_data_t *sst_data_d){
  uint8_t *__restrict__ adc = sst_data_d->adc;
  detId_t *__restrict__ detId = sst_data_d->detId;
  uint16_t *__restrict__ stripId = sst_data_d->stripId;
  fedId_t *__restrict__ fedId = sst_data_d->fedId;
  fedCh_t *__restrict__ fedCh = sst_data_d->fedCh;

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;
  const auto chan = nthreads*bid + tid;

  if (chan < chanlocs->size()) {
    const auto fedid = chanlocs->fedID(chan);
    const auto fedch = chanlocs->fedCh(chan);
    const auto detid = conditions->detID(fedid, fedch);
    const auto ipoff = kStripsPerChannel*conditions->iPair(fedid, fedch);

    const auto data = chanlocs->input(chan);
    const auto len = chanlocs->length(chan);

    if (data != nullptr && len > 0) {
      auto aoff = chanlocs->offset(chan);
      auto choff = chanlocs->inoff(chan);
      const auto end = aoff + len;

      while (aoff < end) {
        stripId[aoff] = invStrip;
        detId[aoff] = invDet;
        adc[aoff] = data[(choff++)^7];
        auto stripIndex = adc[aoff++] + ipoff;

        stripId[aoff] = invStrip;
        detId[aoff] = detid;
        adc[aoff] = data[(choff++)^7];
        const auto groupLength = adc[aoff++];

        for (auto i = 0; i < groupLength; ++i) {
	  fedId[aoff] = fedid;
	  fedCh[aoff] = fedch;
          detId[aoff] = detid;
          stripId[aoff] = stripIndex++;
          adc[aoff++] = data[(choff++)^7];
        }
      }
    }
  }
}

__global__
static void unpackChannelsCalib(const ChanLocStruct* chanlocs, const SiStripConditionsGPU* conditions, calib_data_t *calib_data_d) {
  float *__restrict__ noise = calib_data_d->noise;
  float *__restrict__ gain = calib_data_d->gain;
  bool *__restrict__ bad = calib_data_d->bad;

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;
  const auto chan = nthreads*bid + tid;

  if (chan < chanlocs->size()) {
    const auto fedid = chanlocs->fedID(chan);
    const auto fedch = chanlocs->fedCh(chan);
    const auto detid = conditions->detID(fedid, fedch);
    const auto ipoff = kStripsPerChannel*conditions->iPair(fedid, fedch);

    const auto data = chanlocs->input(chan);
    const auto len = chanlocs->length(chan);

    if (data != nullptr && len > 0) {
      auto aoff = chanlocs->offset(chan);
      auto choff = chanlocs->inoff(chan);
      const auto end = aoff + len;

      while (aoff < end) {
        auto stripIndex = data[(choff++)^7] + ipoff;
	const auto groupLength = data[(choff++)^7];
	aoff=aoff+2;
        for (auto i = 0; i < groupLength; ++i) {
          noise[aoff] = conditions->noise(fedid, fedch, stripIndex);
          gain[aoff]  = conditions->gain(fedid, fedch, stripIndex);
          bad[aoff++]   = conditions->bad(fedid, fedch, stripIndex);
        }
      }
    }
  }
}

__global__
static void setSeedStripsGPU(sst_data_t *sst_data_d, calib_data_t *calib_data_d, const SiStripConditionsGPU * conditions) {
  const int nStrips = sst_data_d->nStrips;
#ifndef USE_TEXTURE
  const uint8_t *__restrict__ adc = sst_data_d->adc;
  const float *__restrict__ noise = calib_data_d->noise;
  const uint16_t *__restrict__ stripId = sst_data_d->stripId;
  const fedId_t *__restrict__ fedId = sst_data_d->fedId;
  const fedCh_t *__restrict__ fedCh = sst_data_d->fedCh;
#endif
  int *__restrict__ seedStripsMask = sst_data_d->seedStripsMask;
  int *__restrict__ seedStripsNCMask = sst_data_d->seedStripsNCMask;

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;
  const float SeedThreshold = 3.0;

  int i = nthreads * bid + tid;
  stripId_t strip;
  float noise_i;
  fedId_t fed;
  fedCh_t channel;
  uint8_t adc_i;

  if (i<nStrips) {
    seedStripsMask[i] = 0;
    seedStripsNCMask[i] = 0;
    strip = stripId[i];
    if (strip != invStrip) {
#ifdef CALIB_1D
      noise_i = NOISE(i);
#else
      fed = fedId[i];
      channel = fedCh[i];
      //    stripId_t strip = stripId[i];
      noise_i = conditions->noise(fed, channel, strip);
#endif
      adc_i = ADC(i);
      seedStripsMask[i] = (adc_i >= static_cast<uint8_t>( noise_i * SeedThreshold)) ? 1:0;
      seedStripsNCMask[i] = seedStripsMask[i];
    }
   }
}

__global__
  static void setNCSeedStripsGPU(sst_data_t *sst_data_d) {
  const int nStrips = sst_data_d->nStrips;
  const detId_t *__restrict__ detId = sst_data_d->detId;
#ifndef USE_TEXTURE
  const uint16_t *__restrict__ stripId = sst_data_d->stripId;
#endif
  const int *__restrict__ seedStripsMask = sst_data_d->seedStripsMask;
  int *__restrict__ seedStripsNCMask = sst_data_d->seedStripsNCMask;

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;

  int i = nthreads * bid + tid;

  if (i>0&&i<nStrips) {
     if (seedStripsMask[i]&&seedStripsMask[i-1]&&(STRIPID(i)-STRIPID(i-1))==1&&(detId[i]==detId[i-1])) seedStripsNCMask[i] = 0;
  }
}

__global__
static void setStripIndexGPU(sst_data_t *sst_data_d) {
  const int nStrips = sst_data_d->nStrips;
  const int *__restrict__ seedStripsNCMask = sst_data_d->seedStripsNCMask;
  const int *__restrict__ prefixSeedStripsNCMask = sst_data_d->prefixSeedStripsNCMask;
  int *__restrict__ seedStripsNCIndex = sst_data_d->seedStripsNCIndex;

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;

  int i = nthreads * bid + tid;
  int index;

  if (i<nStrips) {
    if (seedStripsNCMask[i] == 1) {
      index = prefixSeedStripsNCMask[i];
      seedStripsNCIndex[index] = i;
    }
  }

}

__global__
static void findLeftRightBoundaryGPU(sst_data_t *sst_data_d, calib_data_t *calib_data_d, const SiStripConditionsGPU *conditions, clust_data_t *clust_data_d) {
  const int nStrips = sst_data_d->nStrips;
  const int *__restrict__ seedStripsNCIndex = sst_data_d->seedStripsNCIndex;
  const int nSeedStripsNC = sst_data_d->nSeedStripsNC;
#ifndef USE_TEXTURE
  //#ifdef CALIB_1D
  const float *__restrict__ noise = calib_data_d->noise;
  //#else
  const uint8_t *__restrict__ adc = sst_data_d->adc;
  const uint16_t *__restrict__ stripId = sst_data_d->stripId;
  const detId_t *__restrict__ detId = sst_data_d->detId;
  const fedId_t *__restrict__ fedId = sst_data_d->fedId;
  const fedCh_t *__restrict__ fedCh = sst_data_d->fedCh;
  //#endif
#endif
  int *__restrict__ clusterLastIndexLeft = clust_data_d->clusterLastIndexLeft;
  int *__restrict__ clusterLastIndexRight = clust_data_d->clusterLastIndexRight;
  bool *__restrict__ trueCluster = clust_data_d->trueCluster;

   const uint8_t MaxSequentialHoles = 0;
   const float  ChannelThreshold = 2.0;
   const float ClusterThresholdSquared = 25.0;

   const int tid = threadIdx.x;
   const int bid = blockIdx.x;
   const int nthreads = blockDim.x;

   int index, testIndexLeft, testIndexRight, indexLeft, indexRight, rangeLeft, rangeRight;
   uint8_t testADC;
   float noise_i, testNoise, noiseSquared_i, adcSum_i;
   bool noiseSquaredPass, sameDetLeft, sameDetRight;
   fedId_t fed, testFed;
   fedCh_t channel, testChannel;
   stripId_t strip, testStrip;
   int i = nthreads * bid + tid;

   if (i<nSeedStripsNC) {
     index=seedStripsNCIndex[i];
     indexLeft = index;
     indexRight = index;
#ifdef CALIB_1D
     noise_i = NOISE(index);
#else
     fed = fedId[index];
     channel = fedCh[index];
     strip = stripId[index];
     noise_i = conditions->noise(fed, channel, strip);
#endif
     noiseSquared_i = noise_i*noise_i;
     adcSum_i = static_cast<float>(ADC(index));

     // find left boundary
     testIndexLeft=index-1;
     if (testIndexLeft>=0) {
       rangeLeft = STRIPID(indexLeft)-STRIPID(testIndexLeft)-1;
       sameDetLeft = detId[index] == detId[testIndexLeft];
       while(sameDetLeft&&testIndexLeft>=0&&rangeLeft>=0&&rangeLeft<=MaxSequentialHoles) {
#ifdef CALIB_1D
	 testNoise = NOISE(testIndexLeft);
#else
	 testFed = fedId[testIndexLeft];
	 testChannel = fedCh[testIndexLeft];
	 testStrip = stripId[testIndexLeft];
	 testNoise = conditions->noise(testFed, testChannel, testStrip);
#endif
	 testADC = ADC(testIndexLeft);

	 if (testADC >= static_cast<uint8_t>(testNoise * ChannelThreshold)) {
	   --indexLeft;
	   noiseSquared_i += testNoise*testNoise;
	   adcSum_i += static_cast<float>(testADC);
	 }
	 --testIndexLeft;
	 if (testIndexLeft>=0) {
	   rangeLeft = STRIPID(indexLeft)-STRIPID(testIndexLeft)-1;
	   sameDetLeft = detId[index] == detId[testIndexLeft];
	 }
       }
     }

     // find right boundary
     testIndexRight=index+1;
     if (testIndexRight<nStrips) {
       rangeRight = STRIPID(testIndexRight)-STRIPID(indexRight)-1;
       sameDetRight = detId[index] == detId[testIndexRight];
       while(sameDetRight&&testIndexRight<nStrips&&rangeRight>=0&&rangeRight<=MaxSequentialHoles) {
#ifdef CALIB_1D
	 testNoise = NOISE(testIndexRight);
#else
         testFed = fedId[testIndexRight];
         testChannel = fedCh[testIndexRight];
         testStrip = stripId[testIndexRight];
         testNoise = conditions->noise(testFed, testChannel, testStrip);
#endif
	 testADC = ADC(testIndexRight);
	 if (testADC >= static_cast<uint8_t>(testNoise * ChannelThreshold)) {
	   ++indexRight;
	   noiseSquared_i += testNoise*testNoise;
	   adcSum_i += static_cast<float>(testADC);
	 }
	 ++testIndexRight;
	 if (testIndexRight<nStrips) {
	   rangeRight = STRIPID(testIndexRight)-STRIPID(indexRight)-1;
	   sameDetRight = detId[index] == detId[testIndexRight];
	 }
       }
     }
     noiseSquaredPass = noiseSquared_i*ClusterThresholdSquared <= adcSum_i*adcSum_i;
     trueCluster[i] = noiseSquaredPass;
     clusterLastIndexLeft[i] = indexLeft;
     clusterLastIndexRight[i] = indexRight;

   }
}

__global__
static void checkClusterConditionGPU(sst_data_t *sst_data_d, calib_data_t *calib_data_d, const SiStripConditionsGPU *conditions, clust_data_t *clust_data_d) {
#ifndef USE_TEXTURE
  //#ifdef CALIB_1D
   const float *__restrict__ gain = calib_data_d->gain;
   //#else
   const uint16_t *__restrict__ stripId = sst_data_d->stripId;
   const uint8_t *__restrict__ adc = sst_data_d->adc;
   const fedId_t *__restrict__ fedId = sst_data_d->fedId;
   const fedCh_t *__restrict__ fedCh = sst_data_d->fedCh;
   //#endif
#endif
   const int nSeedStripsNC = sst_data_d->nSeedStripsNC;
   const int *__restrict__ clusterLastIndexLeft = clust_data_d->clusterLastIndexLeft;
   const int *__restrict__ clusterLastIndexRight = clust_data_d->clusterLastIndexRight;
   uint8_t *__restrict__ clusterADCs = clust_data_d->clusterADCs;
   bool *__restrict__ trueCluster = clust_data_d->trueCluster;
   float *__restrict__ barycenter = clust_data_d->barycenter;
   const float minGoodCharge = 1620.0;
   const uint16_t stripIndexMask = 0x7FFF;

   const int tid = threadIdx.x;
   const int bid = blockIdx.x;
   const int nthreads = blockDim.x;

   int i = nthreads * bid + tid;

   int left, right, size, j;
   int charge;
   uint8_t adc_j;
   float gain_j;
   float adcSum=0.0f;
   int sumx=0;
   int suma=0;
   fedId_t fed;
   fedCh_t channel;
   stripId_t strip;

   if (i<nSeedStripsNC) {
     if (trueCluster[i]) {
       left=clusterLastIndexLeft[i];
       right=clusterLastIndexRight[i];
       size=right-left+1;

       if (i>0&&clusterLastIndexLeft[i-1]==left) {
         trueCluster[i] = 0;  // ignore duplicates
       } else {
         for (j=0; j<size; j++){
	   adc_j = ADC(left+j);
#ifdef CALIB_1D
	   gain_j = GAIN(left+j);
#else
	   fed = fedId[left+j];
	   channel = fedCh[left+j];
	   strip = stripId[left+j];
	   gain_j = conditions->gain(fed, channel, strip);
#endif
	   charge = static_cast<int>( static_cast<float>(adc_j)/gain_j + 0.5f );
	   if (adc_j < 254) adc_j = ( charge > 1022 ? 255 : (charge > 253 ? 254 : charge));
	   clusterADCs[j*nSeedStripsNC+i] = adc_j;
	   adcSum += static_cast<float>(adc_j);
	   sumx += j*adc_j;
	   suma += adc_j;
         }
	 barycenter[i] = static_cast<float>(stripId[left] & stripIndexMask) + static_cast<float>(sumx)/static_cast<float>(suma) + 0.5f;
       }
       trueCluster[i] = (adcSum/0.047f) > minGoodCharge;
     }
   }
}

extern "C"
void allocateSSTDataGPU(int max_strips, sst_data_t *sst_data_d, sst_data_t **pt_sst_data_d, gpu_timing_t* gpu_timing,  int dev, hipStream_t stream) {
#ifdef GPU_TIMER
  gpu_timer_start(gpu_timing, stream);
#endif

#ifdef CACHE_ALLOC
  *pt_sst_data_d = (sst_data_t *)cudautils::allocate_device(dev, sizeof(sst_data_t), stream);
  sst_data_d->detId = (detId_t*)cudautils::allocate_device(dev, max_strips*sizeof(detId_t), stream);
  sst_data_d->stripId = (uint16_t *)cudautils::allocate_device(dev, max_strips*sizeof(uint16_t), stream);
  sst_data_d->adc = (uint8_t *)cudautils::allocate_device(dev, max_strips*sizeof(uint8_t), stream);
  sst_data_d->fedId = (fedId_t *)cudautils::allocate_device(dev, max_strips*sizeof(fedId_t), stream);
  sst_data_d->fedCh = (fedCh_t *)cudautils::allocate_device(dev, max_strips*sizeof(fedCh_t), stream);
  sst_data_d->seedStripsMask = (int *)cudautils::allocate_device(dev, 2*max_strips*sizeof(int), stream);
  sst_data_d->prefixSeedStripsNCMask = (int *)cudautils::allocate_device(dev, 2*max_strips*sizeof(int), stream);
#else
  CUDA_RT_CALL(hipMalloc((void **)pt_sst_data_d, sizeof(sst_data_t)));
  CUDA_RT_CALL(hipMalloc((void **)&(sst_data_d->detId), max_strips*sizeof(detId_t)));
  CUDA_RT_CALL(hipMalloc((void **)&(sst_data_d->stripId), 2*max_strips*sizeof(uint16_t)));
  CUDA_RT_CALL(hipMalloc((void **)&(sst_data_d->adc), max_strips*sizeof(uint8_t)));
  CUDA_RT_CALL(hipMalloc((void **)&(sst_data_d->fedId), max_strips*sizeof(fedId_t)));
  CUDA_RT_CALL(hipMalloc((void **)&(sst_data_d->fedCh), max_strips*sizeof(fedCh_t)));
  CUDA_RT_CALL(hipMalloc((void **)&(sst_data_d->seedStripsMask), 2*max_strips*sizeof(int)));
  CUDA_RT_CALL(hipMalloc((void **)&(sst_data_d->prefixSeedStripsNCMask), 2*max_strips*sizeof(int)));
#endif

  sst_data_d->seedStripsNCMask = sst_data_d->seedStripsMask + max_strips;
  sst_data_d->seedStripsNCIndex = sst_data_d->prefixSeedStripsNCMask + max_strips;
  sst_data_d->d_temp_storage=NULL;
  sst_data_d->temp_storage_bytes=0;
  hipcub::DeviceScan::ExclusiveSum(sst_data_d->d_temp_storage, sst_data_d->temp_storage_bytes, sst_data_d->seedStripsNCMask, sst_data_d->prefixSeedStripsNCMask, max_strips);
#ifdef GPU_DEBUG
  std::cout<<"temp_storage_bytes="<<sst_data_d->temp_storage_bytes<<std::endl;
#endif

#ifdef CACHE_ALLOC
  sst_data_d->d_temp_storage = cudautils::allocate_device(dev, sst_data_d->temp_storage_bytes, stream);
#else
  CUDA_RT_CALL(hipMalloc((void **)&(sst_data_d->d_temp_storage), sst_data_d->temp_storage_bytes));
#endif // end CACHE_ALLOC

  CUDA_RT_CALL(hipMemcpyAsync((void *)*pt_sst_data_d, sst_data_d, sizeof(sst_data_t), hipMemcpyHostToDevice, stream));

#ifdef GPU_TIMER
  gpu_timing->memAllocTime = 0;
  gpu_timing->memAllocTime += gpu_timer_measure_end(gpu_timing, stream);
#endif
}

extern "C"
void allocateCalibDataGPU(int max_strips, calib_data_t *calib_data_d, calib_data_t **pt_calib_data_d, gpu_timing_t* gpu_timing, int dev, hipStream_t stream) {
#ifdef GPU_TIMER
  gpu_timer_start(gpu_timing, stream);
#endif

#ifdef CACHE_ALLOC
  *pt_calib_data_d = (calib_data_t *)cudautils::allocate_device(dev, sizeof(calib_data_t), stream);
  calib_data_d->noise = (float *)cudautils::allocate_device(dev, 2*max_strips*sizeof(float), stream);
  calib_data_d->bad = (bool *)cudautils::allocate_device(dev, max_strips*sizeof(bool), stream);
#else
  CUDA_RT_CALL(hipMalloc((void **)pt_calib_data_d, sizeof(calib_data_t)));
  CUDA_RT_CALL(hipMalloc((void **)&(calib_data_d->noise), 2*max_strips*sizeof(float)));
  CUDA_RT_CALL(hipMalloc((void **)&(calib_data_d->bad), max_strips*sizeof(bool)));
#endif
  calib_data_d->gain = calib_data_d->noise + max_strips;
  CUDA_RT_CALL(hipMemcpyAsync((void *)*pt_calib_data_d, calib_data_d, sizeof(calib_data_t), hipMemcpyHostToDevice, stream));
#ifdef GPU_TIMER
  gpu_timing->memAllocTime += gpu_timer_measure_end(gpu_timing, stream);
#endif
}

extern "C"
  void allocateClustDataGPU(int max_strips, clust_data_t *clust_data_d, clust_data_t **pt_clust_data_d, gpu_timing_t *gpu_timing, int dev, hipStream_t stream) {
#ifdef GPU_TIMER
  gpu_timer_start(gpu_timing, stream);
#endif

#ifdef CACHE_ALLOC
  *pt_clust_data_d = (clust_data_t *)cudautils::allocate_device(dev, sizeof(clust_data_t), stream);
  clust_data_d->clusterLastIndexLeft = (int *)cudautils::allocate_device(dev, 2*max_strips*sizeof(int), stream);
  clust_data_d->clusterADCs = (uint8_t *)cudautils::allocate_device(dev, max_strips*256*sizeof(uint8_t), stream);
  clust_data_d->trueCluster = (bool *)cudautils::allocate_device(dev, max_strips*sizeof(bool), stream);
  clust_data_d->barycenter = (float *)cudautils::allocate_device(dev, max_strips*sizeof(float), stream);
#else
  CUDA_RT_CALL(hipMalloc((void **)pt_clust_data_d, sizeof(clust_data_t)));
  CUDA_RT_CALL(hipMalloc((void **)&(clust_data_d->clusterLastIndexLeft), 2*max_strips*sizeof(int)));
  CUDA_RT_CALL(hipMalloc((void **)&(clust_data_d->clusterADCs), max_strips*256*sizeof(uint8_t)));
  CUDA_RT_CALL(hipMalloc((void **)&(clust_data_d->trueCluster), max_strips*sizeof(bool)));
  CUDA_RT_CALL(hipMalloc((void **)&(clust_data_d->barycenter), max_strips*sizeof(float)));
#endif
  clust_data_d->clusterLastIndexRight = clust_data_d->clusterLastIndexLeft + max_strips;
  CUDA_RT_CALL(hipMemcpyAsync((void *)*pt_clust_data_d, clust_data_d, sizeof(clust_data_t), hipMemcpyHostToDevice, stream));

#ifdef GPU_TIMER
  gpu_timing->memAllocTime += gpu_timer_measure_end(gpu_timing, stream);
#endif
}

extern "C"
void freeSSTDataGPU(sst_data_t *sst_data_d, sst_data_t *pt_sst_data_d, gpu_timing_t *gpu_timing, int dev, hipStream_t stream) {
#ifdef GPU_TIMER
  gpu_timer_start(gpu_timing, stream);
#endif

#ifdef CACHE_ALLOC
  cudautils::free_device(dev, pt_sst_data_d);
  cudautils::free_device(dev, sst_data_d->detId);
  cudautils::free_device(dev, sst_data_d->stripId);
  cudautils::free_device(dev, sst_data_d->adc);
  cudautils::free_device(dev, sst_data_d->fedId);
  cudautils::free_device(dev, sst_data_d->fedCh);
  cudautils::free_device(dev, sst_data_d->seedStripsMask);
  cudautils::free_device(dev, sst_data_d->prefixSeedStripsNCMask);
#else
  CUDA_RT_CALL(hipFree(pt_sst_data_d));
  CUDA_RT_CALL(hipFree(sst_data_d->detId));
  CUDA_RT_CALL(hipFree(sst_data_d->stripId));
  CUDA_RT_CALL(hipFree(sst_data_d->adc);
  CUDA_RT_CALL(hipFree(sst_data_d->fedId));
  CUDA_RT_CALL(hipFree(sst_data_d->fedCh));
  CUDA_RT_CALL(hipFree(sst_data_d->seedStripsMask));
  CUDA_RT_CALL(hipFree(sst_data_d->prefixSeedStripsNCMask));
#endif
#if USE_TEXTURE
  hipUnbindTexture(stripIdTexRef);
  hipUnbindTexture(adcTexRef);
#endif
#ifdef GPU_TIMER
  gpu_timing->memFreeTime += gpu_timer_measure_end(gpu_timing, stream);
#endif
}

extern "C"
void freeCalibDataGPU(calib_data_t *calib_data_d, calib_data_t *pt_calib_data_d, gpu_timing_t *gpu_timing, int dev, hipStream_t stream) {
#ifdef GPU_TIMER
  gpu_timer_start(gpu_timing, stream);
#endif

#ifdef CACHE_ALLOC
  cudautils::free_device(dev, pt_calib_data_d);
  cudautils::free_device(dev, calib_data_d->noise);
  cudautils::free_device(dev, calib_data_d->bad);
#else
  CUDA_RT_CALL(hipFree(pt_calib_data_d));
  CUDA_RT_CALL(hipFree(calib_data_d->noise));
  CUDA_RT_CALL(hipFree(calib_data_d->bad));
#endif
#if USE_TEXTURE
  hipUnbindTexture(noiseTexRef);
  hipUnbindTexture(gainTexRef);
#endif
#ifdef GPU_TIMER
  gpu_timing->memFreeTime += gpu_timer_measure_end(gpu_timing, stream);
#endif
}

extern "C"
void freeClustDataGPU(clust_data_t *clust_data_d, clust_data_t *pt_clust_data_d, gpu_timing_t *gpu_timing, int dev, hipStream_t stream) {
#ifdef GPU_TIMER
  gpu_timer_start(gpu_timing, stream);
#endif
#ifdef CACHE_ALLOC
  cudautils::free_device(dev, pt_clust_data_d);
  cudautils::free_device(dev, clust_data_d->clusterLastIndexLeft);
  cudautils::free_device(dev, clust_data_d->clusterADCs);
  cudautils::free_device(dev, clust_data_d->trueCluster);
  cudautils::free_device(dev, clust_data_d->barycenter);
#else
  CUDA_RT_CALL(hipFree(pt_clust_data_d));
  CUDA_RT_CALL(hipFree(clust_data_d->clusterLastIndexLeft));
  CUDA_RT_CALL(hipFree(clust_data_d->clusterADCs));
  CUDA_RT_CALL(hipFree(clust_data_d->trueCluster));
  CUDA_RT_CALL(hipFree(clust_data_d->barycenter));
#endif
#ifdef GPU_TIMER
  gpu_timing->memFreeTime = 0;
  gpu_timing->memFreeTime += gpu_timer_measure_end(gpu_timing, stream);
#endif
}

extern "C"
  void unpackRawDataGPU(const SiStripConditions *conditions, const SiStripConditionsGPU *conditionsGPU, const std::vector<FEDRawData>& fedRawDatav, const std::vector<FEDBuffer>& fedBufferv, const std::vector<fedId_t>& fedIndex, sst_data_t *sst_data_d, sst_data_t *pt_sst_data_d, calib_data_t *calib_data_d, calib_data_t *pt_calib_data_d, const FEDReadoutMode& mode, gpu_timing_t *gpu_timing, hipStream_t stream, SSTorCALIB unpack_option) {

  ChannelLocs chanlocs(conditions->detToFeds().size(), stream);
  ChannelLocsGPU chanlocsGPU(chanlocs.size(), stream);
  std::vector<uint8_t*> inputGPU(chanlocs.size());
  std::vector<size_t> fedRawDataOffsets;
  fedRawDataOffsets.reserve(SiStripConditions::kFedCount);

  auto fedRawDataHost = cudautils::make_host_unique<uint8_t[]>(sst_data_d->totalRawSize, stream);
  auto fedRawDataGPU = cudautils::make_device_unique<uint8_t[]>(sst_data_d->totalRawSize, stream);

  size_t off = 0;
  for (const auto &d : fedRawDatav) {
    memcpy(fedRawDataHost.get() + off, d.get(), d.size());
    fedRawDataOffsets.push_back(off);
    off += d.size();
  }

  //std::cout<<"fedRawData copy size "<<sst_data_d->totalRawSize<<"at stream "<<stream<<std::endl;
  assert(sst_data_d->totalRawSize==off);

#ifdef GPU_TIMER
  gpu_timer_start(gpu_timing, stream);
#endif
  cudautils::copyAsync(fedRawDataGPU, fedRawDataHost, sst_data_d->totalRawSize, stream);
#ifdef GPU_TIMER
  gpu_timing->memTransHDTime += gpu_timer_measure(gpu_timing, stream);
#endif

  const auto& detmap = conditions->detToFeds();
  size_t offset = 0;

  // iterate over the detector in DetID/APVPair order
  // mapping out where the data are
  const uint16_t headerlen = mode == READOUT_MODE_ZERO_SUPPRESSED ? 7 : 2;

  for(size_t i = 0; i < detmap.size(); ++i) {
    const auto& detp = detmap[i];

    auto fedId = detp.fedID();
    auto fedi = fedIndex[fedId-SiStripConditions::kFedFirst];
    if (fedi != invFed) {
      const auto& buffer = fedBufferv[fedi];
      const auto& channel = buffer.channel(detp.fedCh());

      if (channel.length() >= headerlen) {
        chanlocs.setChannelLoc(i, channel.data(), channel.offset()+headerlen, offset, channel.length()-headerlen, detp.fedID(), detp.fedCh());
	inputGPU[i] = fedRawDataGPU.get() + fedRawDataOffsets[fedi] + (channel.data() - fedRawDatav[fedi].get());
        offset += channel.length()-headerlen;
      } else {
        chanlocs.setChannelLoc(i, channel.data(), channel.offset(), offset, channel.length(), detp.fedID(),detp.fedCh());
	inputGPU[i] = fedRawDataGPU.get() + fedRawDataOffsets[fedi] + (channel.data() - fedRawDatav[fedi].get());
        offset += channel.length();
        assert(channel.length() == 0);
      }
    } else {
      chanlocs.setChannelLoc(i, nullptr, 0, 0, 0, invFed, 0);
      std::cout << "Missing fed " << fedi << " for detID " << detp.fedID() << std::endl;
      inputGPU[i] = nullptr;
      exit (1);
    }
  }

  sst_data_d->nStrips = offset;
  //std::cout<<"nStrips "<<offset<<std::endl;

  if (unpack_option == SST)
    CUDA_RT_CALL(hipMemcpyAsync((void *)&(pt_sst_data_d->nStrips), &(sst_data_d->nStrips), sizeof(int), hipMemcpyHostToDevice, stream));

  chanlocsGPU.reset(chanlocs, inputGPU, stream);

#ifdef GPU_TIMER
  gpu_timing->memTransHDTime += gpu_timer_measure(gpu_timing, stream);
#endif

  constexpr int nthreads = 64;
  const auto channels = chanlocs.size();
  const auto nblocks = (channels + nthreads - 1)/nthreads;

  if (unpack_option == SST)
  //std::cout<<"total channels size "<<channels<<std::endl;
    unpackChannelsSST<<<nblocks, nthreads, 0, stream>>>(chanlocsGPU.chanLocStruct(), conditionsGPU, pt_sst_data_d);
  else if (unpack_option == CALIB)
    unpackChannelsCalib<<<nblocks, nthreads, 0, stream>>>(chanlocsGPU.chanLocStruct(), conditionsGPU, pt_calib_data_d);
  else {
    std::cout<<"other unpack option is not available"<<std::endl;
    exit (2);
  }
  //hipStreamSynchronize(stream);

#ifdef GPU_TIMER
  gpu_timing->unpackRawDataTime = gpu_timer_measure_end(gpu_timing, stream);
#endif

#ifdef GPU_DEBUG
  hipStreamSynchronize(stream);
  int nStrips = sst_data_d->nStrips;
  uint16_t *cpu_strip = (uint16_t *)malloc(nStrips*sizeof(uint16_t));
  uint8_t *cpu_adc = (uint8_t *)malloc(nStrips*sizeof(uint8_t));
  fedId_t *cpu_fedid = (fedId_t *)malloc(nStrips*sizeof(fedId_t));

  hipMemcpy((void *)cpu_strip, sst_data_d->stripId, nStrips*sizeof(uint16_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_adc, sst_data_d->adc, nStrips*sizeof(uint8_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_fedid, sst_data_d->fedId, nStrips*sizeof(fedId_t), hipMemcpyDeviceToHost);

  for (int i=0; i<nStrips; i++) {
    std::cout<<" i "<<i<<" cpu fedId "<<cpu_fedid[i]<<" cpu_strip "<<(unsigned int)cpu_strip[i]<<" cpu_adc "<<(unsigned int)cpu_adc[i]<<std::endl;
  }

  free(cpu_strip);
  free(cpu_adc);
  free(cpu_fedid);
#endif

}


extern "C"
void findClusterGPU(sst_data_t *sst_data_d, sst_data_t *pt_sst_data_d, calib_data_t *calib_data_d, calib_data_t *pt_calib_data_d, const SiStripConditionsGPU *conditions, clust_data_t *clust_data_d, clust_data_t *pt_clust_data_d, gpu_timing_t *gpu_timing, hipStream_t stream) {
#ifdef GPU_TIMER
  gpu_timer_start(gpu_timing, stream);
#endif
  int nthreads = 128;
  //int nSeedStripsNC = sst_data_d->nSeedStripsNC;
  int nSeedStripsNC = MAX_SEEDSTRIPS;
  int nblocks = (nSeedStripsNC+nthreads-1)/nthreads;

#ifdef GPU_DEBUG
  int nStrips = sst_data_d->nStrips;
  int *cpu_index = (int *)malloc(nStrips*sizeof(int));
  uint16_t *cpu_strip = (uint16_t *)malloc(nStrips*sizeof(uint16_t));
  uint8_t *cpu_adc = (uint8_t *)malloc(nStrips*sizeof(uint8_t));
  float *cpu_noise = (float *)malloc(nStrips*sizeof(float));

  hipMemcpy((void *)cpu_strip, sst_data_d->stripId, nStrips*sizeof(uint16_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_adc, sst_data_d->adc, nStrips*sizeof(uint8_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_noise, calib_data_d->noise, nStrips*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_index, sst_data_d->seedStripsNCIndex, nStrips*sizeof(int), hipMemcpyDeviceToHost);

  for (int i=0; i<nStrips; i++) {
      std::cout<<" cpu_strip "<<cpu_strip[i]<<" cpu_adc "<<(unsigned int)cpu_adc[i]<<" cpu index "<<cpu_index[i]<<std::endl;
  }

  free(cpu_index);
  free(cpu_strip);
  free(cpu_adc);
  free(cpu_noise);
#endif

  findLeftRightBoundaryGPU<<<nblocks, nthreads, 0, stream>>>(pt_sst_data_d, pt_calib_data_d, conditions, pt_clust_data_d);
  CUDA_RT_CALL(hipGetLastError());

#ifdef GPU_TIMER
  gpu_timing->findBoundaryTime = gpu_timer_measure(gpu_timing, stream);
#endif

  checkClusterConditionGPU<<<nblocks, nthreads, 0, stream>>>(pt_sst_data_d, pt_calib_data_d, conditions, pt_clust_data_d);
  CUDA_RT_CALL(hipGetLastError());

#ifdef GPU_TIMER
  gpu_timing->checkClusterTime = gpu_timer_measure_end(gpu_timing, stream);
#endif

#ifdef GPU_DEBUG
  int *clusterLastIndexLeft = (int *)malloc(nSeedStripsNC*sizeof(int));
  int *clusterLastIndexRight = (int *)malloc(nSeedStripsNC*sizeof(int));
  bool *trueCluster = (bool *)malloc(nSeedStripsNC*sizeof(bool));
  uint8_t *ADCs = (uint8_t*)malloc(nSeedStripsNC*256*sizeof(uint8_t));
  //nSeedStripsNC=sst_data_d->nSeedStripsNC;
  std::cout<<"findClusterGPU"<<"nSeedStripsNC="<<nSeedStripsNC<<std::endl;
  hipMemcpyAsync((void *)clusterLastIndexLeft, clust_data_d->clusterLastIndexLeft, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpyAsync((void *)clusterLastIndexRight, clust_data_d->clusterLastIndexRight, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpyAsync((void *)trueCluster, clust_data_d->trueCluster, nSeedStripsNC*sizeof(bool), hipMemcpyDeviceToHost);
  hipMemcpyAsync((void *)ADCs, clust_data_d->clusterADCs, nSeedStripsNC*256*sizeof(uint8_t), hipMemcpyDeviceToHost);

  hipStreamSynchronize(stream);
  nSeedStripsNC=sst_data_d->nSeedStripsNC;

  for (int i=0; i<nSeedStripsNC; i++) {
    if (trueCluster[i]){
      int left=clusterLastIndexLeft[i];
      int right=clusterLastIndexRight[i];
      std::cout<<"i="<<i<<" left "<<left<<" right "<<right<<" : ";
      int size=right-left+1;
      for (int j=0; j<size; j++){
	std::cout<<(unsigned int)ADCs[j*nSeedStripsNC+i]<<" ";
      }
      std::cout<<std::endl;
    }
  }

  free(clusterLastIndexLeft);
  free(clusterLastIndexRight);
  free(trueCluster);
  free(ADCs);
#endif

}

extern "C"
void setSeedStripsNCIndexGPU(sst_data_t *sst_data_d, sst_data_t *pt_sst_data_d, calib_data_t *calib_data_d, calib_data_t *pt_calib_data_d, const SiStripConditionsGPU *conditions, gpu_timing_t *gpu_timing, hipStream_t stream) {
#ifdef GPU_DEBUG
  hipStreamSynchronize(stream);
  int nStrips = sst_data_d->nStrips;
  uint16_t *cpu_strip = (uint16_t *)malloc(nStrips*sizeof(uint16_t));
  uint8_t *cpu_adc = (uint8_t *)malloc(nStrips*sizeof(uint8_t));
  float *cpu_noise = (float *)malloc(nStrips*sizeof(float));

  hipMemcpy((void *)cpu_strip, sst_data_d->stripId, nStrips*sizeof(uint16_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_adc, sst_data_d->adc, nStrips*sizeof(uint8_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_noise, calib_data_d->noise, nStrips*sizeof(float), hipMemcpyDeviceToHost);

  for (int i=0; i<nStrips; i++) {
    if (i%1000==0)
    std::cout<<" cpu_strip "<<cpu_strip[i]<<" cpu_adc "<<(unsigned int)cpu_adc[i]<<std::endl;
  }

  free(cpu_strip);
  free(cpu_adc);
  free(cpu_noise);
#endif

  int nthreads = 256;
  int nblocks = (sst_data_d->nStrips+nthreads-1)/nthreads;

#ifdef GPU_TIMER
  gpu_timer_start(gpu_timing, stream);
#endif
  CUDA_RT_CALL(hipMemcpyAsync((void *)&(pt_sst_data_d->nStrips), &(sst_data_d->nStrips), sizeof(int), hipMemcpyHostToDevice, stream));
#ifdef GPU_TIMER
  gpu_timing->memTransHDTime += gpu_timer_measure(gpu_timing, stream);
#endif

  //  hipStreamSynchronize(stream);
  //mark seed strips
  setSeedStripsGPU<<<nblocks, nthreads, 0, stream>>>(pt_sst_data_d, pt_calib_data_d, conditions);
  CUDA_RT_CALL(hipGetLastError());
#ifdef GPU_TIMER
  gpu_timing->setSeedStripsTime = gpu_timer_measure(gpu_timing, stream);
#endif

  //mark only non-consecutive seed strips (mask out consecutive seed strips)
  setNCSeedStripsGPU<<<nblocks, nthreads, 0, stream>>>(pt_sst_data_d);
  CUDA_RT_CALL(hipGetLastError());
#ifdef GPU_TIMER
  gpu_timing->setNCSeedStripsTime = gpu_timer_measure(gpu_timing, stream);
#endif
  //std::cout<<"nStrip in setStripIndexGPU is="<<sst_data_d->nStrips<<std::endl;

  hipcub::DeviceScan::ExclusiveSum(sst_data_d->d_temp_storage, sst_data_d->temp_storage_bytes, sst_data_d->seedStripsNCMask, sst_data_d->prefixSeedStripsNCMask, sst_data_d->nStrips, stream);

#ifdef GPU_TIMER
  gpu_timer_measure(gpu_timing, stream);
#endif

  CUDA_RT_CALL(hipMemcpyAsync((void *)&(pt_sst_data_d->nSeedStripsNC), sst_data_d->prefixSeedStripsNCMask+sst_data_d->nStrips-1, sizeof(int), hipMemcpyDeviceToDevice, stream));
#ifdef GPU_TIMER
  gpu_timing->memTransDHTime += gpu_timer_measure(gpu_timing, stream);
#endif

  CUDA_RT_CALL(hipMemcpyAsync((void *)&(sst_data_d->nSeedStripsNC), &(pt_sst_data_d->nSeedStripsNC), sizeof(int), hipMemcpyDeviceToHost, stream));

#ifdef GPU_TIMER
  gpu_timing->memTransHDTime += gpu_timer_measure(gpu_timing, stream);
#endif

#ifdef GPU_DEBUG
  hipStreamSynchronize(stream);
  if (sst_data_d->nSeedStripsNC==0) {
    std::cout<<"nSeedStripsNC in setStripIndexGPU is="<<sst_data_d->nSeedStripsNC<<std::endl;
    exit(2);
  }
#endif

  setStripIndexGPU<<<nblocks, nthreads, 0, stream>>>(pt_sst_data_d);
  CUDA_RT_CALL(hipGetLastError());

#ifdef GPU_TIMER
  gpu_timing->setStripIndexTime = gpu_timer_measure_end(gpu_timing, stream);
#endif

#ifdef GPU_DEBUG
  int *cpu_mask = (int *)malloc(nStrips*sizeof(int));
  int *cpu_prefix= (int *)malloc(nStrips*sizeof(int));
  int *cpu_index = (int *)malloc(nStrips*sizeof(int));

  hipMemcpy((void *)cpu_mask, sst_data_d->seedStripsNCMask, nStrips*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_prefix, sst_data_d->prefixSeedStripsNCMask, nStrips*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_index, sst_data_d->seedStripsNCIndex, nStrips*sizeof(int), hipMemcpyDeviceToHost);

  for (int i=0; i<nStrips; i++) {
    std::cout<<" i "<<i<<" mask "<<cpu_mask[i]<<" prefix "<<cpu_prefix[i]<<" index "<<cpu_index[i]<<std::endl;
  }

  free(cpu_mask);
  free(cpu_prefix);
  free(cpu_index);

  hipMemcpy((void *)&(sst_data_d->nSeedStripsNC), &(pt_sst_data_d->nSeedStripsNC), sizeof(int), hipMemcpyDeviceToHost);
  std::cout<<"nStrips="<<nStrips<<"nSeedStripsNC="<<sst_data_d->nSeedStripsNC<<"temp_storage_bytes="<<sst_data_d->temp_storage_bytes<<std::endl;
#endif
}


extern "C"
void cpyGPUToCPU(sst_data_t * sst_data_d, sst_data_t *pt_sst_data_d, clust_data_t *clust_data, clust_data_t *clust_data_d, gpu_timing_t *gpu_timing, hipStream_t stream) {
  //  hipDeviceSynchronize();
  //hipMemcpyAsync((void *)&(sst_data_d->nSeedStripsNC), &(pt_sst_data_d->nSeedStripsNC), sizeof(int), hipMemcpyDeviceToHost, stream);
  //hipMemcpy((void *)&(sst_data_d->nSeedStripsNC), &(pt_sst_data_d->nSeedStripsNC), sizeof(int), hipMemcpyDeviceToHost);
  //hipStreamSynchronize(stream);

  int nSeedStripsNC = 150000;
  //std::cout<<"cpyGPUtoCPU Event="<<event<<"offset="<<offset<<"nSeedStripsNC="<<nSeedStripsNC<<std::endl;
#ifdef GPU_TIMER
  gpu_timer_start(gpu_timing, stream);
#endif
  CUDA_RT_CALL(hipMemcpyAsync((void *)(clust_data->clusterLastIndexLeft), clust_data_d->clusterLastIndexLeft, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost, stream));
  CUDA_RT_CALL(hipMemcpyAsync((void *)(clust_data->clusterLastIndexRight), clust_data_d->clusterLastIndexRight, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost, stream));
#ifdef COPY_ADC
  CUDA_RT_CALL(hipMemcpyAsync((void *)(clust_data->clusterADCs), clust_data_d->clusterADCs, nSeedStripsNC*256*sizeof(uint8_t), hipMemcpyDeviceToHost, stream));
#endif
  CUDA_RT_CALL(hipMemcpyAsync((void *)(clust_data->trueCluster), clust_data_d->trueCluster, nSeedStripsNC*sizeof(bool), hipMemcpyDeviceToHost, stream));
  CUDA_RT_CALL(hipMemcpyAsync((void *)(clust_data->barycenter), clust_data_d->barycenter, nSeedStripsNC*sizeof(float), hipMemcpyDeviceToHost, stream));
  //CUDA_RT_CALL(hipMemcpyAsync((void *)&(sst_data_d->nSeedStripsNC), &(pt_sst_data_d->nSeedStripsNC), sizeof(int), hipMemcpyDeviceToHost, stream));
  //CUDA_RT_CALL(hipStreamSynchronize(stream));
  //CUDA_RT_CALL(hipMemcpy((void *)&(sst_data_d->nSeedStripsNC), &(pt_sst_data_d->nSeedStripsNC), sizeof(int), hipMemcpyDeviceToHost));
#ifdef GPU_TIMER
  gpu_timing->memTransDHTime = gpu_timer_measure_end(gpu_timing, stream);
#endif
}

extern "C"
void cpyCalibDataToGPU(int max_strips, calib_data_t *calib_data, calib_data_t *calib_data_d, gpu_timing_t *gpu_timing, hipStream_t stream) {
#ifdef GPU_TIMER
  gpu_timer_start(gpu_timing, stream);
#endif
  CUDA_RT_CALL(hipMemcpy((void *)calib_data_d->noise, calib_data->noise, max_strips*sizeof(float), hipMemcpyHostToDevice));
  CUDA_RT_CALL(hipMemcpy((void *)calib_data_d->gain, calib_data->gain, max_strips*sizeof(float), hipMemcpyHostToDevice));
#if USE_TEXTURE
  hipBindTexture(0, noiseTexRef, (void *)calib_data_d->noise, max_strips*sizeof(float));
  hipBindTexture(0, gainTexRef, (void *)calib_data_d->gain, max_strips*sizeof(float));
#endif
#ifdef GPU_TIMER
  gpu_timing->memTransHDTime = gpu_timer_measure_end(gpu_timing, stream);
#endif
}

extern "C"
void cpySSTDataToGPU(sst_data_t *sst_data, sst_data_t *sst_data_d, gpu_timing_t *gpu_timing, hipStream_t stream) {
#ifdef GPU_TIMER
  gpu_timer_start(gpu_timing, stream);
#endif
  sst_data_d->nStrips = sst_data->nStrips;
  int nStrips = sst_data_d->nStrips;
  std::cout<<"cpySSTDataToGPU nStrips="<<nStrips<<std::endl;

  CUDA_RT_CALL(hipMemcpyAsync((void *)sst_data_d->stripId, sst_data->stripId, nStrips*sizeof(uint16_t), hipMemcpyHostToDevice, stream));
  CUDA_RT_CALL(hipMemcpyAsync((void *)sst_data_d->detId, sst_data->detId, nStrips*sizeof(detId_t), hipMemcpyHostToDevice, stream));
  CUDA_RT_CALL(hipMemcpyAsync((void *)sst_data_d->fedId, sst_data->fedId, nStrips*sizeof(fedId_t), hipMemcpyHostToDevice, stream));
  CUDA_RT_CALL(hipMemcpyAsync((void *)sst_data_d->fedCh, sst_data->fedCh, nStrips*sizeof(fedCh_t), hipMemcpyHostToDevice, stream));
  CUDA_RT_CALL(hipMemcpyAsync((void *)sst_data_d->adc, sst_data->adc, nStrips*sizeof(uint8_t), hipMemcpyHostToDevice, stream));
#if USE_TEXTURE
  hipBindTexture(0, stripIdTexRef, (void *)sst_data_d->stripId, nStrips*sizeof(uint16_t));
  hipBindTexture(0, adcTexRef, (void *)sst_data_d->adc, nStrips*sizeof(uint8_t));
#endif
#ifdef GPU_TIMER
  gpu_timing->memTransHDTime = gpu_timer_measure_end(gpu_timing, stream);
#endif
}
