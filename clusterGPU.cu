#include "hip/hip_runtime.h"
#include "clusterGPU.cuh"
#include <hipcub/hipcub.hpp>
#include <stdio.h>

#if USE_TEXTURE
texture<float, 1, hipReadModeElementType> noiseTexRef;
texture<float, 1, hipReadModeElementType> gainTexRef;
texture<uint16_t, 1, hipReadModeElementType> stripIdTexRef;
texture<uint16_t, 1, hipReadModeElementType> adcTexRef;

static __inline__ __device__ float fetch_noise(int i)
{
  return tex1Dfetch(noiseTexRef, i);
}
static __inline__ __device__ float fetch_gain(int i)
{
  return tex1Dfetch(gainTexRef, i);
}
static __inline__ __device__ uint16_t fetch_stripId(int i)
{
  return tex1Dfetch(stripIdTexRef, i);
}

static __inline__ __device__ uint16_t fetch_adc(int i)
{
  return tex1Dfetch(adcTexRef, i);
}
#define NOISE(i) (fetch_noise(i))
#define GAIN(i) (fetch_gain(i))
#define STRIPID(i) (fetch_stripId(i))
#define ADC(i) (fetch_adc(i))
#else
#define NOISE(i) (noise[i])
#define GAIN(i) (gain[i])
#define STRIPID(i) (stripId[i])
#define ADC(i) (adc[i])
#endif

static void gpu_timer_start(gpu_timing_t *gpu_timing) {
  hipEventCreate(&gpu_timing->start);
  hipEventCreate(&gpu_timing->stop);
  hipEventRecord(gpu_timing->start,0);
}

static float gpu_timer_measure(gpu_timing_t *gpu_timing) {
  float elapsedTime;
  hipEventRecord(gpu_timing->stop,0);
  hipEventSynchronize(gpu_timing->stop);
  hipEventElapsedTime(&elapsedTime, gpu_timing->start, gpu_timing->stop);
  hipEventRecord(gpu_timing->start,0);

  return elapsedTime/1000;
}

static float gpu_timer_measure_end(gpu_timing_t *gpu_timing) {
  float elapsedTime;
  hipEventRecord(gpu_timing->stop,0);
  hipEventSynchronize(gpu_timing->stop);
  hipEventElapsedTime(&elapsedTime, gpu_timing->start,gpu_timing->stop);

  hipEventDestroy(gpu_timing->start);
  hipEventDestroy(gpu_timing->stop);
  return elapsedTime/1000;
}

__global__
static void setSeedStripsGPU(int nStrips, sst_data_t *sst_data_d, calib_data_t *calib_data_d) {
#ifndef USE_TEXTURE
  const uint16_t *__restrict__ adc = sst_data_d->adc;
  const float *__restrict__ noise = calib_data_d->noise;
#endif
  int *__restrict__ seedStripsMask = sst_data_d->seedStripsMask;
  int *__restrict__ seedStripsNCMask = sst_data_d->seedStripsNCMask;

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;
  const float SeedThreshold = 3.0;

  int i = nthreads * bid + tid;

  if (i<nStrips) {
    seedStripsMask[i] = 0;
    seedStripsNCMask[i] = 0;
    float noise_i = NOISE(i);
    uint8_t adc_i = static_cast<uint8_t>(ADC(i));
    seedStripsMask[i] = (adc_i >= static_cast<uint8_t>( noise_i * SeedThreshold)) ? 1:0;
    seedStripsNCMask[i] = seedStripsMask[i];
  }
}

__global__
static void setNCSeedStripsGPU(int nStrips, sst_data_t *sst_data_d) {
#ifndef USE_TEXTURE
  const uint16_t *__restrict__ stripId = sst_data_d->stripId;
#endif
  const int *__restrict__ seedStripsMask = sst_data_d->seedStripsMask;
  int *__restrict__ seedStripsNCMask = sst_data_d->seedStripsNCMask;

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;

  int i = nthreads * bid + tid;

  if (i>0&&i<nStrips) {
    if (seedStripsMask[i]&&seedStripsMask[i-1]&&(STRIPID(i)-STRIPID(i-1))==1) seedStripsNCMask[i] = 0;
  }
}

__global__
static void setStripIndexGPU(int nStrips, sst_data_t *sst_data_d) {
  const int *__restrict__ seedStripsNCMask = sst_data_d->seedStripsNCMask;
  const int *__restrict__ prefixSeedStripsNCMask = sst_data_d->prefixSeedStripsNCMask;
  int *__restrict__ seedStripsNCIndex = sst_data_d->seedStripsNCIndex;

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;

  int i = nthreads * bid + tid;

  if (i<nStrips) {
    if (seedStripsNCMask[i] == 1) {
      int index = prefixSeedStripsNCMask[i];
      seedStripsNCIndex[index] = i;
    }
  }

}

__global__
static void findLeftRightBoundaryGPU(int nSeedStripsNC, int nStrips, sst_data_t *sst_data_d, calib_data_t *calib_data_d, clust_data_t *clust_data_d) {
  const int *__restrict__ seedStripsNCIndex = sst_data_d->seedStripsNCIndex;
#ifndef USE_TEXTURE
  const uint16_t *__restrict__ stripId = sst_data_d->stripId;
  const uint16_t *__restrict__ adc = sst_data_d->adc;
  const float *__restrict__ noise = calib_data_d->noise;
#endif
  int *__restrict__ clusterLastIndexLeft = clust_data_d->clusterLastIndexLeft;
  int *__restrict__ clusterLastIndexRight = clust_data_d->clusterLastIndexRight;
  bool *__restrict__ trueCluster = clust_data_d->trueCluster;

   const uint8_t MaxSequentialHoles = 0;
   const float  ChannelThreshold = 2.0;
   const float minGoodCharge = 1620.0;
   const float ClusterThresholdSquared = 25.0;

   const int tid = threadIdx.x;
   const int bid = blockIdx.x;
   const int nthreads = blockDim.x;

   int index, testIndexLeft, testIndexRight, indexLeft, indexRight, rangeLeft, rangeRight;
   uint8_t testADC;
   float noise_i, testNoise, noiseSquared_i, adcSum_i;
   bool noiseSquaredPass, chargePerCMPass;
   int i = nthreads * bid + tid;

   if (i<nSeedStripsNC) {

     index=seedStripsNCIndex[i];
     indexLeft = index;
     indexRight = index;
     noise_i = NOISE(index);
     noiseSquared_i = noise_i*noise_i;
     adcSum_i = static_cast<float>(ADC(index));

     // find left boundary
     testIndexLeft=index-1;
     rangeLeft = STRIPID(indexLeft)-STRIPID(testIndexLeft)-1;

     while(testIndexLeft>=0&&rangeLeft>=0&&rangeLeft<=MaxSequentialHoles) {
       testNoise = NOISE(testIndexLeft);
       testADC = static_cast<uint8_t>(ADC(testIndexLeft));

       if (testADC > static_cast<uint8_t>(testNoise * ChannelThreshold)) {
	 --indexLeft;
	 noiseSquared_i += testNoise*testNoise;
	 adcSum_i += static_cast<float>(testADC);
       }
       --testIndexLeft;
       rangeLeft = STRIPID(indexLeft)-STRIPID(testIndexLeft)-1;
     }

     // find right boundary
     testIndexRight=index+1;
     rangeRight = STRIPID(testIndexRight)-STRIPID(indexRight)-1;

     while(testIndexRight<nStrips&&rangeRight>=0&&rangeRight<=MaxSequentialHoles) {
       testNoise = NOISE(testIndexRight);
       testADC = static_cast<uint8_t>(ADC(testIndexRight));
       if (testADC > static_cast<uint8_t>(testNoise * ChannelThreshold)) {
	 ++indexRight;
	 noiseSquared_i += testNoise*testNoise;
	 adcSum_i += static_cast<float>(testADC);
       }
       ++testIndexRight;
       rangeRight = STRIPID(testIndexRight)-STRIPID(indexRight)-1;
     }

     noiseSquaredPass = noiseSquared_i*ClusterThresholdSquared <= adcSum_i*adcSum_i;
     chargePerCMPass = adcSum_i/0.047f > minGoodCharge;

     trueCluster[i] = noiseSquaredPass&&chargePerCMPass;
     clusterLastIndexLeft[i] = indexLeft;
     clusterLastIndexRight[i] = indexRight;
   }
}

__global__
static void checkClusterConditionGPU(int nSeedStripsNC,  sst_data_t *sst_data_d, calib_data_t *calib_data_d, clust_data_t *clust_data_d) {
#ifndef USE_TEXTURE
   const uint16_t *__restrict__ stripId = sst_data_d->stripId;
   const uint16_t *__restrict__ adc = sst_data_d->adc;
   const float *__restrict__ noise = calib_data_d->noise;
#endif
   int *__restrict__ clusterLastIndexLeft = clust_data_d->clusterLastIndexLeft;
   int *__restrict__ clusterLastIndexRight = clust_data_d->clusterLastIndexRight;
   uint8_t *__restrict__ clusterADCs = clust_data_d->clusterADCs;
   bool *__restrict__ trueCluster = clust_data_d->trueCluster;

   const int tid = threadIdx.x;
   const int bid = blockIdx.x;
   const int nthreads = blockDim.x;

   int i = nthreads * bid + tid;

   int left, right, size, j;
   int charge;
   uint16_t adc_j;
   float gain_j;

   if (i<nSeedStripsNC) {
     if (trueCluster[i]) {
       left=clusterLastIndexLeft[i];
       right=clusterLastIndexRight[i];
       size=right-left+1;

       for (j=0; j<size; j++){
	 adc_j = ADC(left+j);
	 gain_j = GAIN(left+j);
	 charge = static_cast<int>( static_cast<float>(adc_j)/gain_j + 0.5f );
	 if (adc_j < 254) adc_j = ( charge > 1022 ? 255 : (charge > 253 ? 254 : charge));
	 clusterADCs[j*nSeedStripsNC+i] = adc_j;
       }
     }
   }
}

extern "C"
void allocateSSTDataGPU(int nStrips, sst_data_t *sst_data_d, sst_data_t **pt_sst_data_d) {
  hipMalloc((void **)pt_sst_data_d, sizeof(sst_data_t));
  hipMalloc((void **)&(sst_data_d->stripId), nStrips*sizeof(uint16_t));
  hipMalloc((void **)&(sst_data_d->adc), nStrips*sizeof(uint16_t));
  hipMalloc((void **)&(sst_data_d->seedStripsMask), nStrips*sizeof(int));
  hipMalloc((void **)&(sst_data_d->seedStripsNCMask), nStrips*sizeof(int));
  hipMalloc((void **)&(sst_data_d->prefixSeedStripsNCMask), nStrips*sizeof(int));
  hipMalloc((void **)&(sst_data_d->seedStripsNCIndex), nStrips*sizeof(int));
  hipMemcpy((void *)*pt_sst_data_d, sst_data_d, sizeof(sst_data_t), hipMemcpyHostToDevice);
}

extern "C"
void allocateCalibDataGPU(int nStrips, calib_data_t *calib_data_d, calib_data_t **pt_calib_data_d) {
  hipMalloc((void **)pt_calib_data_d, sizeof(calib_data_t));
  hipMalloc((void **)&(calib_data_d->noise), nStrips*sizeof(float));
  hipMalloc((void **)&(calib_data_d->gain), nStrips*sizeof(float));
  hipMalloc((void **)&(calib_data_d->bad), nStrips*sizeof(bool));
  hipMemcpy((void *)*pt_calib_data_d, calib_data_d, sizeof(calib_data_t), hipMemcpyHostToDevice);
}

extern "C"
void allocateClustDataGPU(int nSeedStripsNC, clust_data_t *clust_data_d, clust_data_t **pt_clust_data_d) {
  hipMalloc((void **)pt_clust_data_d, sizeof(clust_data_t));
  hipMalloc((void **)&(clust_data_d->clusterLastIndexLeft), nSeedStripsNC*sizeof(int));
  hipMalloc((void **)&(clust_data_d->clusterLastIndexRight), nSeedStripsNC*sizeof(int));
  hipMalloc((void **)&(clust_data_d->clusterADCs), nSeedStripsNC*256*sizeof(uint8_t));
  hipMalloc((void **)&(clust_data_d->trueCluster), nSeedStripsNC*sizeof(bool));
  hipMemcpy((void *)*pt_clust_data_d, clust_data_d, sizeof(clust_data_t), hipMemcpyHostToDevice);
}

extern "C"
void freeGPUMem(sst_data_t *sst_data_d, sst_data_t *pt_sst_data_d, calib_data_t *calib_data_d, calib_data_t *pt_calib_data_d, clust_data_t *clust_data_d, clust_data_t *pt_clust_data_d) {
  hipFree(pt_sst_data_d);
  hipFree(pt_calib_data_d);
  hipFree(pt_clust_data_d);
  hipFree(sst_data_d->stripId);
  hipFree(sst_data_d->adc);
  hipFree(sst_data_d->seedStripsMask);
  hipFree(sst_data_d->seedStripsNCMask);
  hipFree(sst_data_d->prefixSeedStripsNCMask);
  hipFree(sst_data_d->seedStripsNCIndex);
  hipFree(calib_data_d->noise);
  hipFree(calib_data_d->gain);
  hipFree(calib_data_d->bad);
#if USE_TEXTURE
  hipUnbindTexture(stripIdTexRef);
  hipUnbindTexture(adcTexRef);
  hipUnbindTexture(noiseTexRef);
  hipUnbindTexture(gainTexRef);
#endif
  hipFree(clust_data_d->clusterLastIndexLeft);
  hipFree(clust_data_d->clusterLastIndexRight);
  hipFree(clust_data_d->clusterADCs);
  hipFree(clust_data_d->trueCluster);
}

extern "C"
void findClusterGPU(int nSeedStripsNC, int nStrips, sst_data_t *sst_data_d, sst_data_t *pt_sst_data_d, calib_data_t *calib_data_d, calib_data_t *pt_calib_data_d, clust_data_t *clust_data_d, clust_data_t *pt_clust_data_d, gpu_timing_t *gpu_timing) {
  gpu_timer_start(gpu_timing);
  int nthreads = 128;
  int nblocks = (nSeedStripsNC+nthreads-1)/nthreads;

#ifdef GPU_DEBUG
  int *cpu_index = (int *)malloc(nSeedStripsNC*sizeof(int));
  uint16_t *cpu_strip = (uint16_t *)malloc(nStrips*sizeof(uint16_t));
  uint16_t *cpu_adc = (uint16_t *)malloc(nStrips*sizeof(uint16_t));
  float *cpu_noise = (float *)malloc(nStrips*sizeof(float));

  hipMemcpy((void *)cpu_strip, sst_data_d->stripId, nStrips*sizeof(uint16_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_adc, sst_data_d->adc, nStrips*sizeof(uint16_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_noise, calib_data_d->noise, nStrips*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_index, sst_data_d->seedStripsNCIndex, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);

  for (int i=0; i<nStrips; i++) {
    std::cout<<" cpu_strip "<<cpu_strip[i]<<" cpu_adc "<<cpu_adc[i]<<" cpu_noise "<<cpu_noise[i]<<" cpu index "<<cpu_index[i]<<std::endl;
  }

  free(cpu_index);
  free(cpu_strip);
  free(cpu_adc);
  free(cpu_noise);
#endif

  findLeftRightBoundaryGPU<<<nblocks, nthreads>>>(nSeedStripsNC, nStrips, pt_sst_data_d, pt_calib_data_d, pt_clust_data_d);

  gpu_timing->findBoundaryTime = gpu_timer_measure(gpu_timing);

  checkClusterConditionGPU<<<nblocks, nthreads>>>(nSeedStripsNC, pt_sst_data_d, pt_calib_data_d, pt_clust_data_d);

  gpu_timing->checkClusterTime = gpu_timer_measure_end(gpu_timing);

#ifdef GPU_DEBUG
  int *clusterLastIndexLeft = (int *)malloc(nSeedStripsNC*sizeof(int));
  int *clusterLastIndexRight = (int *)malloc(nSeedStripsNC*sizeof(int));
  bool *trueCluster = (bool *)malloc(nSeedStripsNC*sizeof(bool));
  uint8_t *ADCs = (uint8_t*)malloc(nSeedStripsNC*256*sizeof(uint8_t));

  hipMemcpy((void *)clusterLastIndexLeft, clust_data_d->clusterLastIndexLeft, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)clusterLastIndexRight, clust_data_d->clusterLastIndexRight, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)trueCluster, clust_data_d->trueCluster, nSeedStripsNC*sizeof(bool), hipMemcpyDeviceToHost);
  hipMemcpy((void *)ADCs, clust_data_d->clusterADCs, nSeedStripsNC*256*sizeof(uint8_t), hipMemcpyDeviceToHost);

  for (int i=0; i<nSeedStripsNC; i++) {
    if (trueCluster[i]){
      int left=clusterLastIndexLeft[i];
      int right=clusterLastIndexRight[i];
      std::cout<<" left "<<left<<" right "<<right<<" : ";
      int size=right-left+1;
      for (int j=0; j<size; j++){
	std::cout<<(int)ADCs[j*nSeedStripsNC+i]<<" ";
      }
      std::cout<<std::endl;
    }
  }

  free(clusterLastIndexLeft);
  free(clusterLastIndexRight);
  free(trueCluster);
  free(ADCs);
#endif

}

extern "C"
int setSeedStripsNCIndexGPU(int nStrips, sst_data_t *sst_data_d, sst_data_t *pt_sst_data_d, calib_data_t *calib_data_d, calib_data_t *pt_calib_data_d, gpu_timing_t *gpu_timing) {
#ifdef GPU_DEBUG
  uint16_t *cpu_strip = (uint16_t *)malloc(nStrips*sizeof(uint16_t));
  uint16_t *cpu_adc = (uint16_t *)malloc(nStrips*sizeof(uint16_t));
  float *cpu_noise = (float *)malloc(nStrips*sizeof(float));

  hipMemcpy((void *)cpu_strip, sst_data_d->stripId, nStrips*sizeof(uint16_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_adc, sst_data_d->adc, nStrips*sizeof(uint16_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_noise, calib_data_d->noise, nStrips*sizeof(float), hipMemcpyDeviceToHost);

  for (int i=0; i<nStrips; i++) {
    std::cout<<" cpu_strip "<<cpu_strip[i]<<" cpu_adc "<<cpu_adc[i]<<" cpu_noise "<<cpu_noise[i]<<std::endl;
  }

  free(cpu_strip);
  free(cpu_adc);
  free(cpu_noise);
#endif
  int nSeedStripsNC;
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;

  int nthreads = 256;
  int nblocks = (nStrips+nthreads-1)/nthreads;

  gpu_timer_start(gpu_timing);
  //mark seed strips
  setSeedStripsGPU<<<nblocks, nthreads>>>(nStrips, pt_sst_data_d, pt_calib_data_d);
  gpu_timing->setSeedStripsTime = gpu_timer_measure(gpu_timing);

  //mark only non-consecutive seed strips (mask out consecutive seed strips)
  setNCSeedStripsGPU<<<nblocks, nthreads>>>(nStrips, pt_sst_data_d);
  gpu_timing->setNCSeedStripsTime = gpu_timer_measure(gpu_timing);

  // set index for non-consecutive seed strips
  hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, sst_data_d->seedStripsNCMask, sst_data_d->prefixSeedStripsNCMask, nStrips);

  hipMalloc(&d_temp_storage, temp_storage_bytes);

  hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, sst_data_d->seedStripsNCMask, sst_data_d->prefixSeedStripsNCMask, nStrips);

  hipMemcpy((void *)&nSeedStripsNC, sst_data_d->prefixSeedStripsNCMask+nStrips-1, sizeof(int), hipMemcpyDeviceToHost);

  setStripIndexGPU<<<nblocks, nthreads>>>(nStrips, pt_sst_data_d);

  hipFree(d_temp_storage);

  gpu_timing->setStripIndexTime = gpu_timer_measure_end(gpu_timing);

#ifdef GPU_DEBUG
  int *cpu_mask = (int *)malloc(nStrips*sizeof(int));
  int *cpu_prefix= (int *)malloc(nStrips*sizeof(int));
  int *cpu_index = (int *)malloc(nStrips*sizeof(int));

  hipMemcpy((void *)cpu_mask, sst_data_d->seedStripsNCMask, nStrips*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_prefix, sst_data_d->prefixSeedStripsNCMask, nStrips*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_index, sst_data_d->seedStripsNCIndex, nStrips*sizeof(int), hipMemcpyDeviceToHost);

  for (int i=0; i<nStrips; i++) {
    std::cout<<" i "<<i<<" mask "<<cpu_mask[i]<<" prefix "<<cpu_prefix[i]<<" index "<<cpu_index[i]<<std::endl;
  }

  free(cpu_mask);
  free(cpu_prefix);
  free(cpu_index);

  std::cout<<"nStrips="<<nStrips<<"nSeedStripsNC="<<nSeedStripsNC<<"temp_storage_bytes="<<temp_storage_bytes<<std::endl;
#endif

  return nSeedStripsNC;
}


extern "C"
void cpyGPUToCPU(int nSeedStripsNC, clust_data_t *clust_data, clust_data_t *clust_data_d) {
  hipMemcpy((void *)clust_data->clusterLastIndexLeft, clust_data_d->clusterLastIndexLeft, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)clust_data->clusterLastIndexRight, clust_data_d->clusterLastIndexRight, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)clust_data->clusterADCs, clust_data_d->clusterADCs, nSeedStripsNC*256*sizeof(uint8_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)clust_data->trueCluster, clust_data_d->trueCluster, nSeedStripsNC*sizeof(bool), hipMemcpyDeviceToHost);
}

extern "C"
void cpyCPUToGPU(int nStrips, sst_data_t *sst_data, sst_data_t *sst_data_d, calib_data_t *calib_data, calib_data_t *calib_data_d, gpu_timing_t *gpu_timing) {
  gpu_timer_start(gpu_timing);
  hipMemcpy((void *)sst_data_d->stripId, sst_data->stripId, nStrips*sizeof(uint16_t), hipMemcpyHostToDevice);
  hipMemcpy((void *)sst_data_d->adc, sst_data->adc, nStrips*sizeof(uint16_t), hipMemcpyHostToDevice);
  hipMemcpy((void *)calib_data_d->noise, calib_data->noise, nStrips*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy((void *)calib_data_d->gain, calib_data->gain, nStrips*sizeof(float), hipMemcpyHostToDevice);
#if USE_TEXTURE
  hipBindTexture(0, stripIdTexRef, (void *)sst_data_d->stripId, nStrips*sizeof(uint16_t));
  hipBindTexture(0, adcTexRef, (void *)sst_data_d->adc, nStrips*sizeof(uint16_t));
  hipBindTexture(0, noiseTexRef, (void *)calib_data_d->noise, nStrips*sizeof(float));
  hipBindTexture(0, gainTexRef, (void *)calib_data_d->gain, nStrips*sizeof(float));
#endif
  gpu_timing->memTransferTime = gpu_timer_measure_end(gpu_timing);
}
