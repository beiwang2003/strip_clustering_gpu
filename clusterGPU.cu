#include "hip/hip_runtime.h"
#include "clusterGPU.cuh"
#include <hipcub/hipcub.hpp>
#include <stdio.h>

#if USE_TEXTURE
texture<float, 1, hipReadModeElementType> noiseTexRef;
texture<float, 1, hipReadModeElementType> gainTexRef;
texture<uint16_t, 1, hipReadModeElementType> stripIdTexRef;
texture<uint16_t, 1, hipReadModeElementType> adcTexRef;

static __inline__ __device__ float fetch_noise(int i)
{
  return tex1Dfetch(noiseTexRef, i);
}
static __inline__ __device__ float fetch_gain(int i)
{
  return tex1Dfetch(gainTexRef, i);
}
static __inline__ __device__ uint16_t fetch_stripId(int i)
{
  return tex1Dfetch(stripIdTexRef, i);
}

static __inline__ __device__ uint16_t fetch_adc(int i)
{
  return tex1Dfetch(adcTexRef, i);
}
#define NOISE(i) (fetch_noise(i))
#define GAIN(i) (fetch_gain(i))
#define STRIPID(i) (fetch_stripId(i))
#define ADC(i) (fetch_adc(i))
#else
#define NOISE(i) (noise_d[i])
#define GAIN(i) (gain_d[i])
#define STRIPID(i) (stripId_d[i])
#define ADC(i) (adc_d[i])
#endif

static void gpu_timer_start(gpu_timing_t *gpu_timing)
{
  hipEventCreate(&gpu_timing->start);
  hipEventCreate(&gpu_timing->stop);
  hipEventRecord(gpu_timing->start,0);
}

static float gpu_timer_measure(gpu_timing_t *gpu_timing)
{
  float elapsedTime;
  hipEventRecord(gpu_timing->stop,0);
  hipEventSynchronize(gpu_timing->stop);
  hipEventElapsedTime(&elapsedTime, gpu_timing->start, gpu_timing->stop);
  hipEventRecord(gpu_timing->start,0);

  return elapsedTime/1000;
}

static float gpu_timer_measure_end(gpu_timing_t *gpu_timing)
{
  float elapsedTime;
  hipEventRecord(gpu_timing->stop,0);
  hipEventSynchronize(gpu_timing->stop);
  hipEventElapsedTime(&elapsedTime, gpu_timing->start,gpu_timing->stop);

  hipEventDestroy(gpu_timing->start);
  hipEventDestroy(gpu_timing->stop);
  return elapsedTime/1000;
}


__global__
static void setSeedStripsGPU(int nStrips, float *noise_d, uint16_t *adc_d, int *seedStripMask_d, int *seedStripNCMask_d)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;
  float SeedThreshold = 3.0;

  int i = nthreads * bid + tid;

  if (i<nStrips) {
    seedStripMask_d[i] = 0;
    seedStripNCMask_d[i] = 0;
    float noise_i = NOISE(i);
    uint8_t adc_i = static_cast<uint8_t>(ADC(i));
    seedStripMask_d[i] = (adc_i >= static_cast<uint8_t>( noise_i * SeedThreshold)) ? 1:0;
    seedStripNCMask_d[i] = seedStripMask_d[i];
  }
}

__global__
static void setNCSeedStripsGPU(int nStrips, uint16_t *stripId_d, int *seedStripMask_d, int *seedStripNCMask_d)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;

  int i = nthreads * bid + tid;

  if (i>0&&i<nStrips) {
    if (seedStripMask_d[i]&&seedStripMask_d[i-1]&&(STRIPID(i)-STRIPID(i-1))==1) seedStripNCMask_d[i] = 0;
  }
}

__global__
static void setStripIndexGPU(int nStrips, int *seedStripNCMask_d, int *prefixSeedStripNCMask_d, int *seedStripsNCIndex_d)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int nthreads = blockDim.x;

  int i = nthreads * bid + tid;

  if (i<nStrips) {
    if (seedStripNCMask_d[i] == 1) {
      int index = prefixSeedStripNCMask_d[i];
      seedStripsNCIndex_d[index] = i;
    }
  }

}

__global__
static void findLeftRightBoundaryGPU(int nSeedStripsNC, int nStrips, int *clusterLastIndexLeft_d, int *clusterLastIndexRight_d, int *seedStripsNCIndex_d, uint16_t *stripId_d, uint16_t *adc_d, float *noise_d, bool *trueCluster_d)
{
   const uint8_t MaxSequentialHoles = 0;
   const float  ChannelThreshold = 2.0;
   const float minGoodCharge = 1620.0;
   const float ClusterThresholdSquared = 25.0;

   const int tid = threadIdx.x;
   const int bid = blockIdx.x;
   const int nthreads = blockDim.x;

   int index, testIndexLeft, testIndexRight, indexLeft, indexRight, rangeLeft, rangeRight;
   uint8_t testADC;
   float noise_i, testNoise, noiseSquared_i, adcSum_i;
   bool noiseSquaredPass, chargePerCMPass;
   int i = nthreads * bid + tid;

   if (i<nSeedStripsNC) {

     index=seedStripsNCIndex_d[i];
     indexLeft = index;
     indexRight = index;
     noise_i = NOISE(index);
     noiseSquared_i = noise_i*noise_i;
     adcSum_i = static_cast<float>(ADC(index));

     // find left boundary
     testIndexLeft=index-1;
     rangeLeft = STRIPID(indexLeft)-STRIPID(testIndexLeft)-1;

     while(testIndexLeft>=0&&rangeLeft>=0&&rangeLeft<=MaxSequentialHoles) {
       testNoise = NOISE(testIndexLeft);
       testADC = static_cast<uint8_t>(ADC(testIndexLeft));

       if (testADC > static_cast<uint8_t>(testNoise * ChannelThreshold)) {
	 --indexLeft;
	 noiseSquared_i += testNoise*testNoise;
	 adcSum_i += static_cast<float>(testADC);
       }
       --testIndexLeft;
       rangeLeft = STRIPID(indexLeft)-STRIPID(testIndexLeft)-1;
     }

     // find right boundary
     testIndexRight=index+1;
     rangeRight = STRIPID(testIndexRight)-STRIPID(indexRight)-1;

     while(testIndexRight<nStrips&&rangeRight>=0&&rangeRight<=MaxSequentialHoles) {
       testNoise = NOISE(testIndexRight);
       testADC = static_cast<uint8_t>(ADC(testIndexRight));
       if (testADC > static_cast<uint8_t>(testNoise * ChannelThreshold)) {
	 ++indexRight;
	 noiseSquared_i += testNoise*testNoise;
	 adcSum_i += static_cast<float>(testADC);
       }
       ++testIndexRight;
       rangeRight = STRIPID(testIndexRight)-STRIPID(indexRight)-1;
     }

     noiseSquaredPass = noiseSquared_i*ClusterThresholdSquared <= adcSum_i*adcSum_i;
     chargePerCMPass = adcSum_i/0.047f > minGoodCharge;

     trueCluster_d[i] = noiseSquaredPass&chargePerCMPass;
     clusterLastIndexLeft_d[i] = indexLeft;
     clusterLastIndexRight_d[i] = indexRight;
   }
}

__global__
static void checkClusterConditionGPU(int nSeedStripsNC, int *clusterLastIndexLeft_d, int *clusterLastIndexRight_d, uint16_t *adc_d, float * gain_d, bool *trueCluster_d, uint8_t *clusterADCs_d)
{
   const int tid = threadIdx.x;
   const int bid = blockIdx.x;
   const int nthreads = blockDim.x;

   int i = nthreads * bid + tid;

   int left, right, size, j;
   int charge;
   uint16_t adc_j;
   float gain_j;

   if (i<nSeedStripsNC) {
     if (trueCluster_d[i]) {
       left=clusterLastIndexLeft_d[i];
       right=clusterLastIndexRight_d[i];
       size=right-left+1;

       for (j=0; j<size; j++){
	 adc_j = ADC(left+j);
	 gain_j = GAIN(left+j);
	 charge = static_cast<int>( static_cast<float>(adc_j)/gain_j + 0.5f );
	 if (adc_j < 254) adc_j = ( charge > 1022 ? 255 : (charge > 253 ? 254 : charge));
	 clusterADCs_d[j*nSeedStripsNC+i] = adc_j;
       }
     }
   }
}

extern "C"
void allocateMemAllStripsGPU(int max_strips, uint16_t **stripId_d_pt, uint16_t **adc_d_pt, float **noise_d_pt, float **gain_d_pt, int **seedStripsNCIndex_d_pt, int **seedStripsMask_d_pt, int **seedStripsNCMask_d_pt, int **prefixSeedStripsNCMask_d_pt)
{
  hipMalloc(stripId_d_pt, max_strips*sizeof(uint16_t));
  hipMalloc(adc_d_pt, max_strips*sizeof(uint16_t));
  hipMalloc(noise_d_pt, max_strips*sizeof(float));
  hipMalloc(gain_d_pt, max_strips*sizeof(float));
  hipMalloc(seedStripsNCIndex_d_pt, max_strips*sizeof(int));

  hipMalloc(seedStripsMask_d_pt, max_strips*sizeof(int));
  hipMalloc(seedStripsNCMask_d_pt, max_strips*sizeof(int));
  hipMalloc(prefixSeedStripsNCMask_d_pt, max_strips*sizeof(int));
}

extern "C"
void allocateMemNCSeedStripsGPU(int nSeedStripsNC, int **clusterLastIndexLeft_d_pt, int **clusterLastIndexRight_d_pt, uint8_t **clusterADCs_d_pt, bool **trueCluster_d_pt)
{
  hipMalloc(clusterLastIndexLeft_d_pt, 2*nSeedStripsNC*sizeof(int));
  *clusterLastIndexRight_d_pt = *clusterLastIndexLeft_d_pt + nSeedStripsNC;
  hipMalloc(clusterADCs_d_pt, nSeedStripsNC*256*sizeof(uint8_t));
  hipMalloc(trueCluster_d_pt, nSeedStripsNC*sizeof(bool));
}

extern "C"
void freeGPUMem(uint16_t *stripId_d, uint16_t *adc_d, float *noise_d, float *gain_d, int *seedStripNCIndex_d, int *seedStripsMask_d, int *seedStripsNCMask_d, int *prefixSeedStripsNCMask_d, int *clusterLastIndexLeft_d, uint8_t *clusterADCs_d, bool *trueCluster_d)
{
   hipFree(stripId_d);
   hipFree(adc_d);
   hipFree(noise_d);
   hipFree(gain_d);
#if USE_TEXTURE
   hipUnbindTexture(stripIdTexRef);
   hipUnbindTexture(adcTexRef);
   hipUnbindTexture(noiseTexRef);
   hipUnbindTexture(gainTexRef);
#endif
   hipFree(seedStripsMask_d);
   hipFree(seedStripsNCMask_d);
   hipFree(prefixSeedStripsNCMask_d);
   hipFree(seedStripNCIndex_d);
   hipFree(clusterLastIndexLeft_d);
   hipFree(clusterADCs_d);
   hipFree(trueCluster_d);
}

extern "C"
void  findClusterGPU(int nSeedStripsNC, int nStrips, int *clusterLastIndexLeft_d,  int *clusterLastIndexRight_d, int *seedStripsNCIndex_d, uint16_t *stripId_d, uint16_t *adc_d, float *noise_d, float *gain_d, bool *trueCluster_d, uint8_t *clusterADCs_d, gpu_timing_t *gpu_timing)
{
  gpu_timer_start(gpu_timing);
  int nthreads = 128;
  int nblocks = (nSeedStripsNC+nthreads-1)/nthreads;

#ifdef GPU_DEBUG
  int *cpu_index = (int *)malloc(nSeedStripsNC*sizeof(int));
  uint16_t *cpu_strip = (uint16_t *)malloc(nStrips*sizeof(uint16_t));
  uint16_t *cpu_adc = (uint16_t *)malloc(nStrips*sizeof(uint16_t));
  float *cpu_noise = (float *)malloc(nStrips*sizeof(float));

  hipMemcpy((void *)cpu_strip, stripId_d, nStrips*sizeof(uint16_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_adc, adc_d, nStrips*sizeof(uint16_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_noise, noise_d, nStrips*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_index, seedStripsNCIndex_d, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);

  for (int i=0; i<nStrips; i++) {
    std::cout<<" cpu_strip "<<cpu_strip[i]<<" cpu_adc "<<cpu_adc[i]<<" cpu_noise "<<cpu_noise[i]<<" cpu index "<<cpu_index[i]<<std::endl;
  }

  free(cpu_index);
  free(cpu_strip);
  free(cpu_adc);
  free(cpu_noise);
#endif

  findLeftRightBoundaryGPU<<<nblocks, nthreads>>>(nSeedStripsNC, nStrips, clusterLastIndexLeft_d, clusterLastIndexRight_d, seedStripsNCIndex_d, stripId_d, adc_d, noise_d, trueCluster_d);

  gpu_timing->findBoundaryTime = gpu_timer_measure(gpu_timing);

  checkClusterConditionGPU<<<nblocks, nthreads>>>(nSeedStripsNC, clusterLastIndexLeft_d, clusterLastIndexRight_d, adc_d, gain_d, trueCluster_d, clusterADCs_d);

  gpu_timing->checkClusterTime = gpu_timer_measure_end(gpu_timing);

#ifdef GPU_DEBUG
  int *clusterLastIndexLeft = (int *)malloc(nSeedStripsNC*sizeof(int));
  int *clusterLastIndexRight = (int *)malloc(nSeedStripsNC*sizeof(int));
  bool *trueCluster = (bool *)malloc(nSeedStripsNC*sizeof(bool));
  uint8_t *ADCs = (uint8_t*)malloc(nSeedStripsNC*256*sizeof(uint8_t));

  hipMemcpy((void *)clusterLastIndexLeft, clusterLastIndexLeft_d, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)clusterLastIndexRight, clusterLastIndexRight_d, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)trueCluster, trueCluster_d, nSeedStripsNC*sizeof(bool), hipMemcpyDeviceToHost);
  hipMemcpy((void *)ADCs, clusterADCs_d, nSeedStripsNC*256*sizeof(uint8_t), hipMemcpyDeviceToHost);

  for (int i=0; i<nSeedStripsNC; i++) {
    if (trueCluster[i]){
      int left=clusterLastIndexLeft[i];
      int right=clusterLastIndexRight[i];
      std::cout<<" left "<<left<<" right "<<right<<" : ";
      int size=right-left+1;
      for (int j=0; j<size; j++){
	std::cout<<(int)ADCs[j*nSeedStripsNC+i]<<" ";
      }
      std::cout<<std::endl;
    }
  }

  free(clusterLastIndexLeft);
  free(clusterLastIndexRight);
  free(trueCluster);
  free(ADCs);
#endif

}

extern "C"
int setSeedStripsNCIndexGPU(int nStrips, uint16_t *stripId_d, uint16_t *adc_d, float *noise_d, int *seedStripsNCIndex_d, int *seedStripsMask_d, int *seedStripsNCMask_d, int *prefixSeedStripsNCMask_d, gpu_timing_t *gpu_timing){
  int nSeedStripsNC;

  gpu_timer_start(gpu_timing);

  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;

#ifdef GPU_DEBUG
  int *cpu_mask = (int *)malloc(nStrips*sizeof(int));
  int *cpu_prefix= (int *)malloc(nStrips*sizeof(int));
  int *cpu_index = (int *)malloc(nStrips*sizeof(int));
#endif

  int nthreads = 256;
  int nblocks = (nStrips+nthreads-1)/nthreads;

  // mark seed strips
  setSeedStripsGPU<<<nblocks, nthreads>>>(nStrips, noise_d, adc_d, seedStripsMask_d, seedStripsNCMask_d);
  gpu_timing->setSeedStripsTime = gpu_timer_measure(gpu_timing);

  // mark only non-consecutive seed strips (mask out consecutive seed strips)
  setNCSeedStripsGPU<<<nblocks, nthreads>>>(nStrips, stripId_d, seedStripsMask_d, seedStripsNCMask_d);
  gpu_timing->setNCSeedStripsTime = gpu_timer_measure(gpu_timing);

  // set index for non-consecutive seed strips
  hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, seedStripsNCMask_d, prefixSeedStripsNCMask_d, nStrips);

  hipMalloc(&d_temp_storage, temp_storage_bytes);

  hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, seedStripsNCMask_d, prefixSeedStripsNCMask_d, nStrips);

  hipMemcpy((void *)&nSeedStripsNC, prefixSeedStripsNCMask_d+nStrips-1, sizeof(int), hipMemcpyDeviceToHost);

  setStripIndexGPU<<<nblocks, nthreads>>>(nStrips, seedStripsNCMask_d, prefixSeedStripsNCMask_d, seedStripsNCIndex_d);

  hipFree(d_temp_storage);

#ifdef GPU_DEBUG
  hipMemcpy((void *)cpu_mask, seedStripsNCMask_d, nStrips*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_prefix, prefixSeedStripsNCMask_d, nStrips*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)cpu_index, seedStripsNCIndex_d, nStrips*sizeof(int), hipMemcpyDeviceToHost);

  for (int i=0; i<nStrips; i++) {
    std::cout<<" i "<<i<<" mask "<<cpu_mask[i]<<" prefix "<<cpu_prefix[i]<<" index "<<cpu_index[i]<<std::endl;
  }

  free(cpu_mask);
  free(cpu_prefix);
  free(cpu_index);

  std::cout<<"nStrips="<<nStrips<<"nSeedStripsNC="<<nSeedStripsNC<<"temp_storage_bytes="<<temp_storage_bytes<<std::endl;
#endif

  gpu_timing->setStripIndexTime = gpu_timer_measure_end(gpu_timing);

  return nSeedStripsNC;
}

extern "C"
void cpyGPUToCPU(int nSeedStripsNC, int *clusterLastIndexLeft_d, int *clusterLastIndexLeft, int *clusterLastIndexRight_d, int *clusterLastIndexRight, uint8_t *clusterADCs_d, uint8_t *clusterADCs, bool *trueCluster_d, bool *trueCluster) {
  hipMemcpy((void *)clusterLastIndexLeft, clusterLastIndexLeft_d, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)clusterLastIndexRight, clusterLastIndexRight_d, nSeedStripsNC*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)clusterADCs, clusterADCs_d, nSeedStripsNC*256*sizeof(uint8_t), hipMemcpyDeviceToHost);
  hipMemcpy((void *)trueCluster, trueCluster_d, nSeedStripsNC*sizeof(bool), hipMemcpyDeviceToHost);
}

extern "C"
void cpyCPUToGPU(int nStrips, uint16_t *stripId_d, uint16_t *stripId, uint16_t *adc_d,  uint16_t *adc, float *noise_d, float *noise, float *gain_d, float *gain, gpu_timing_t *gpu_timing) {
  gpu_timer_start(gpu_timing);
  hipMemcpy((void *)stripId_d, stripId, nStrips*sizeof(uint16_t), hipMemcpyHostToDevice);
  hipMemcpy((void *)adc_d, adc, nStrips*sizeof(uint16_t), hipMemcpyHostToDevice);
  hipMemcpy((void *)noise_d, noise, nStrips*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy((void *)gain_d, gain, nStrips*sizeof(float), hipMemcpyHostToDevice);
#if USE_TEXTURE
  hipBindTexture(0, stripIdTexRef, (void *)stripId_d, nStrips*sizeof(uint16_t));
  hipBindTexture(0, adcTexRef, (void *)adc_d, nStrips*sizeof(uint16_t));
  hipBindTexture(0, noiseTexRef, (void *)noise_d, nStrips*sizeof(float));
  hipBindTexture(0, gainTexRef, (void *)gain_d, nStrips*sizeof(float));
#endif
  gpu_timing->memTransferTime = gpu_timer_measure_end(gpu_timing);
}
